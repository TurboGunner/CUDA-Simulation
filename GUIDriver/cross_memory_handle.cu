#include "cuda_interop_helper.cuh"

CrossMemoryHandle::CrossMemoryHandle(const size_t& size_in, const size_t& type_size_in) {
	size = size_in;
	type_size = type_size_in;
}

VkDeviceSize CrossMemoryHandle::TotalAllocationSize() const {
	return size * type_size;
}

hipError_t CrossMemoryHandle::AllocateCudaMemory() {
	hipError_t cuda_status = hipHostMalloc(&cuda_host_ptr, TotalAllocationSize());
	cuda_status = hipMalloc(&cuda_device_ptr, TotalAllocationSize());

	return cuda_status;
}

hipError_t CrossMemoryHandle::DeallocateCudaMemory() {
	hipError_t cuda_status = hipHostFree(cuda_host_ptr);
	cuda_status = hipFree(cuda_device_ptr);

	return cuda_status;
}