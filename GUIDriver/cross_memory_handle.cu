#include "cross_memory_handle.cuh"

CrossMemoryHandle::CrossMemoryHandle(const size_t& size_in, const size_t& type_size_in, const bool& host_inclusive_in) {
	size = size_in;
	type_size = type_size_in;
	host_inclusive = host_inclusive_in;
}

CrossMemoryHandle& CrossMemoryHandle::operator=(const CrossMemoryHandle& src_handle) {
	if (this == &src_handle) {
		return *this;
	}
	size = src_handle.size;
	type_size = src_handle.type_size;
	host_inclusive = src_handle.host_inclusive;

	buffer = src_handle.buffer;
	buffer_memory = src_handle.buffer_memory;

	if (host_inclusive) {
		cuda_host_ptr = src_handle.cuda_host_ptr;
	}
	cuda_device_ptr = src_handle.cuda_device_ptr;
	vulkan_ptr = src_handle.vulkan_ptr;

	cuda_handle = src_handle.cuda_handle;
	shareable_handle = src_handle.shareable_handle;

	granularity_size = src_handle.granularity_size;
}

VkDeviceSize CrossMemoryHandle::TotalAllocationSize() const {
	return size * type_size;
}

hipError_t CrossMemoryHandle::AllocateCudaMemory() {
	hipError_t cuda_status = hipMalloc(&cuda_device_ptr, TotalAllocationSize());
	if (host_inclusive) {
		cuda_status = hipHostMalloc(&cuda_host_ptr, TotalAllocationSize()); //Allocates page-locked memory
	}

	return cuda_status;
}

hipError_t CrossMemoryHandle::DeallocateCudaMemory() {
	hipError_t cuda_status = hipFree(cuda_device_ptr);
	CudaExceptionHandler(cuda_status, "FreeCUDADevice");
	if (host_inclusive) {
		cuda_status = hipHostFree(cuda_host_ptr); //Uses hipHostFree, as it is page-locked memory
		CudaExceptionHandler(cuda_status, "FreeCUDAHostPageLocked");
	}

	return cuda_status;
}