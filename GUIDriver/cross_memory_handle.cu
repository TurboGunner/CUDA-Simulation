#include "cuda_interop_helper.cuh"

CrossMemoryHandle::CrossMemoryHandle(hipMemGenericAllocationHandle_t cuda_handle_in, ShareableHandle shareable_handle_in, const size_t& size_in, const size_t& type_size_in) {
	cuda_handle = cuda_handle_in;
	shareable_handle = shareable_handle_in;

	size = size_in;
	type_size = type_size_in;
}

VkDeviceSize CrossMemoryHandle::TotalAllocationSize() const {
	return size * type_size;
}

hipError_t CrossMemoryHandle::AllocateCudaMemory() {
	hipError_t cuda_status = hipHostMalloc(&cuda_host_ptr, TotalAllocationSize());
	cuda_status = hipMalloc(&cuda_device_ptr, TotalAllocationSize());

	return cuda_status;
}

hipError_t CrossMemoryHandle::DeallocateCudaMemory() {
	hipError_t cuda_status = hipHostFree(cuda_host_ptr);
	cuda_status = hipFree(cuda_device_ptr);

	return cuda_status;
}