#include "cuda_interop_helper.cuh"

#ifdef _WIN64
#include <VersionHelpers.h>
#include <winternl.h>
#include <sddl.h>
#endif

CudaInterop::CudaInterop(VkDevice& device_in, VkPhysicalDevice& phys_device_in) {
    device_ = device_in;
    phys_device_ = phys_device_in;

#ifdef _WIN64
    os_ = IsWindows8OrGreater() ? WINDOWS_MODERN : WINDOWS_OLD;
#else
    os_ = LINUX;
#endif
    if (os_ != LINUX) {
        ipc_handle_type_flag_ = hipMemHandleTypeWin32;
    }
    else {
        ipc_handle_type_flag_ = hipMemHandleTypePosixFileDescriptor;
    }

    hipError_t cuda_status = hipGetDeviceCount(&device_count_);

    if (device_count_ > 1) {
        ProgramLog::OutputLine("Warning: There are multiple CUDA devices!");
    }

    cuda_status = hipGetDevice(&cuda_device_);
}

void CudaInterop::GetDefaultSecurityDescriptor(hipMemAllocationProp* prop) {
    if (os_ == LINUX) {
        return;
    }
    static const char sddl[] = "D:P(OA;;GARCSDWDWOCCDCLCSWLODTWPRPCRFA;;;WD)";
    static OBJECT_ATTRIBUTES obj_attributes;
    static bool obj_attributes_configured = false;

    if (!obj_attributes_configured) {
        PSECURITY_DESCRIPTOR security_descriptor;
        BOOL result = ConvertStringSecurityDescriptorToSecurityDescriptorA(sddl, SDDL_REVISION_1, &security_descriptor, NULL); //NOTE
        if (result == 0) {
            ProgramLog::OutputLine("IPC failure: GetDefaultSecurityDescriptor Failed! (%d)\n", GetLastError());
        }

        InitializeObjectAttributes(&obj_attributes, nullptr, 0, nullptr, security_descriptor); //NOTE

        obj_attributes_configured = true;
    }
    prop->win32HandleMetaData = &obj_attributes;
}

size_t CudaInterop::RoundWarpGranularity(const size_t& size, const size_t& granularity) {
    return ((size + granularity - 1) / granularity) * granularity;
}

void CudaInterop::CalculateTotalMemorySize(const size_t& granularity) {
    total_alloc_size_ = 0;

    for (auto& mem_handle : cross_memory_handles_) {
        size_t current_granularity_size = RoundWarpGranularity(mem_handle.TotalAllocationSize(), granularity);
        total_alloc_size_ += current_granularity_size;
        mem_handle.granularity_size = current_granularity_size;
    }
}

void CudaInterop::AddMemoryHandle(const size_t& size, const size_t& type_size) {  //NOTE: ALLOCATE STRUCT WITH SIZE
    //WIP

    CrossMemoryHandle position_handle(size, type_size);
    cross_memory_handles_.push_back(position_handle);
}

hipError_t CudaInterop::CreateStream(const unsigned int& flags) {
    return hipStreamCreateWithFlags(&cuda_stream_, flags);
}

void CudaInterop::MemoryAllocationProp() {
    current_alloc_prop_.type = hipMemAllocationTypePinned;

    current_alloc_prop_.location.type = hipMemLocationTypeDevice;
    current_alloc_prop_.location.id = cuda_device_;

    current_alloc_prop_.win32HandleMetaData = nullptr;
    current_alloc_prop_.requestedHandleTypes = ipc_handle_type_flag_;

    if (os_ != LINUX) {
        GetDefaultSecurityDescriptor(&current_alloc_prop_);
    }
}

void CudaInterop::MemoryAccessDescriptor() {
    access_descriptor_ = {};
    access_descriptor_.location.id = cuda_device_;
    access_descriptor_.location.type = hipMemLocationTypeDevice;
    access_descriptor_.flags = hipMemAccessFlagsProtReadWrite;
}

hipError_t CudaInterop::SimulationSetupAllocations() {
    hipDeviceptr_t d_ptr = 0U;
    size_t granularity = 0;

    hipError_t cuda_result;

    MemoryAllocationProp();

    cuda_result = hipMemGetAllocationGranularity(&granularity, &current_alloc_prop_, hipMemAllocationGranularityMinimum);
    DriverLog(cuda_result, "Allocation Granularity");

    CalculateTotalMemorySize(granularity);

    cuda_result = hipMemAddressReserve(&d_ptr, total_alloc_size_, granularity, 0U, 0);
    DriverLog(cuda_result, "MemAddressReserve");

    ProgramLog::OutputLine("Granularity: " + std::to_string(granularity) + "\n");

    cuda_result = hipMemCreate(&cross_memory_handles_[0].cuda_handle, cross_memory_handles_[0].granularity_size, &current_alloc_prop_, 0);
    DriverLog(cuda_result, "MemCreate");

    cuda_result = hipMemExportToShareableHandle((void*)&cross_memory_handles_[0].shareable_handle, cross_memory_handles_[0].cuda_handle, ipc_handle_type_flag_, 0);
    DriverLog(cuda_result, "ExportToShareableHandle");

    hipDeviceptr_t va_position = d_ptr; //NOTE: When having other pointers, this will adding the offsets in order to properly account for fitting into the contiguous VA range.
    cross_memory_handles_[0].vulkan_ptr = (void*)va_position;

    cuda_result = hipMemMap(va_position, cross_memory_handles_[0].granularity_size, 0, cross_memory_handles_[0].cuda_handle, 0);
    DriverLog(cuda_result, "MapMemory");

    cuda_result = hipMemRelease(cross_memory_handles_[0].cuda_handle);
    DriverLog(cuda_result, "ReleaseMemory");

    MemoryAccessDescriptor();

    cuda_result = hipMemSetAccess(d_ptr, total_alloc_size_, &access_descriptor_, 1); //Adds read-write access to the whole VA range.
    DriverLog(cuda_result, "SetMemoryAccess");

    return cuda_result;
}

hipError_t CudaInterop::Clean() {
    hipError_t cuda_result;
    for (const auto& mem_handle : cross_memory_handles_) { //Ensures that all allocations are mapped before attempting to unmap memory
        if (!mem_handle.vulkan_ptr) {
            DriverLog(cuda_result, "Clean");
            return cuda_result;
        }
    }

    IPCCloseShareableHandle(cross_memory_handles_[0].shareable_handle);

    cuda_result = hipMemAddressFree((hipDeviceptr_t) cross_memory_handles_[0].vulkan_ptr, total_alloc_size_);
    DriverLog(cuda_result, "VulkanPtrCUDAFree");

    return cuda_result;
}

hipError_t CudaInterop::CleanSynchronization() {
    hipError_t cuda_status = hipSuccess;

    if (cuda_stream_) {
        cuda_status = hipStreamSynchronize(cuda_stream_);
        cuda_status = hipStreamDestroy(cuda_stream_);
    }

    cuda_status = hipDestroyExternalSemaphore(cuda_wait_semaphore_);
    CudaExceptionHandler(cuda_status, "DestroyExternalSemaphoreWait");
    cuda_status = hipDestroyExternalSemaphore(cuda_signal_semaphore_);
    CudaExceptionHandler(cuda_status, "DestroyExternalSemaphoreSignal");

    for (const auto& mem_handle : cross_memory_handles_) {
        vkDestroyBuffer(device_, mem_handle.buffer, nullptr);
        vkFreeMemory(device_, mem_handle.buffer_memory, nullptr);
    }

    return cuda_status;
}

int CudaInterop::IPCCloseShareableHandle(ShareableHandle sh_handle) {
    return CloseHandle(sh_handle);
}

hipError_t CudaInterop::InitializeCudaInterop(VkSemaphore& wait_semaphore, VkSemaphore& signal_semaphore) {
    hipError_t cuda_status = hipSuccess;
    VkResult vulkan_status = VK_SUCCESS;

    VkDeviceSize alloc_size = cross_memory_handles_[0].TotalAllocationSize();
    auto mem_handle_type = GetPlatformMemoryHandle();
    void* mem_handle = (void*) (uintptr_t) &cross_memory_handles_[0].shareable_handle;

    VkBuffer& buffer = cross_memory_handles_[0].buffer;
    VkDeviceMemory& buffer_memory = cross_memory_handles_[0].buffer_memory;

    vulkan_status = ImportExternalBuffer(mem_handle, mem_handle_type, alloc_size, VK_BUFFER_USAGE_TRANSFER_DST_BIT | VK_BUFFER_USAGE_VERTEX_BUFFER_BIT, VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT, buffer, buffer_memory);

    if (vulkan_status != VK_SUCCESS) {
        ProgramLog::OutputLine("Importing external buffer failed in InitializeCudaInterop!");
    }

    ProgramLog::OutputLine("Buffer Memory Size: " + std::to_string(alloc_size));

    auto mem_semaphore_type = GetPlatformSemaphoreHandle();

    vulkan_status = CreateExternalSemaphore(wait_semaphore, mem_semaphore_type);
    VulkanExceptionHandler(vulkan_status, "Failure creating wait semaphore in InitializeCudaInterop!");
    vulkan_status = CreateExternalSemaphore(signal_semaphore, mem_semaphore_type);
    VulkanExceptionHandler(vulkan_status, "Failure creating signal semaphore in InitializeCudaInterop!");

    cuda_status = ImportCudaExternalSemaphore(cuda_wait_semaphore_, wait_semaphore, mem_semaphore_type);
    CudaExceptionHandler(cuda_status, "ImportCUDAExternalSemaphoreWait");
    cuda_status = ImportCudaExternalSemaphore(cuda_signal_semaphore_, signal_semaphore, mem_semaphore_type);
    CudaExceptionHandler(cuda_status, "ImportCUDAExternalSemaphoreSignal");

    return cuda_status;
}

void CudaInterop::InteropExtensions() {
    interop_extensions_.push_back(VK_KHR_EXTERNAL_MEMORY_CAPABILITIES_EXTENSION_NAME);

    interop_extensions_.push_back(VK_KHR_EXTERNAL_SEMAPHORE_CAPABILITIES_EXTENSION_NAME);
    interop_extensions_.push_back(VK_KHR_EXTERNAL_FENCE_CAPABILITIES_EXTENSION_NAME);

    interop_extensions_.push_back(VK_KHR_GET_PHYSICAL_DEVICE_PROPERTIES_2_EXTENSION_NAME);
}

void CudaInterop::InteropDeviceExtensions() {
    interop_device_extensions_.push_back(VK_KHR_EXTERNAL_MEMORY_EXTENSION_NAME);
    interop_device_extensions_.push_back(VK_KHR_EXTERNAL_SEMAPHORE_EXTENSION_NAME);

    if (os_ != LINUX) {
        interop_device_extensions_.push_back(VK_KHR_EXTERNAL_MEMORY_WIN32_EXTENSION_NAME);
        interop_device_extensions_.push_back(VK_KHR_EXTERNAL_SEMAPHORE_WIN32_EXTENSION_NAME);
    }
    else {
        interop_device_extensions_.push_back(VK_KHR_EXTERNAL_MEMORY_FD_EXTENSION_NAME);
        interop_device_extensions_.push_back(VK_KHR_EXTERNAL_SEMAPHORE_FD_EXTENSION_NAME);
    }
}

bool CudaInterop::IsVkPhysicalDeviceUUID(void* uuid) {
    return !memcmp((void*) vk_device_uuid_, uuid, (size_t) VK_UUID_SIZE);
}

void CudaInterop::PopulateCommandBuffer(VkCommandBuffer& command_buffer) {
    vkCmdDraw(command_buffer, cross_memory_handles_[0].size, 1, 0, 0);
}