#include "cuda_interop_helper.cuh"

CudaInterop::CudaInterop(VkDevice& device_in, VkPhysicalDevice& phys_device_in) {
    device_ = device_in;
    phys_device_ = phys_device_in;

#ifdef _WIN64
    os_ = IsWindows8OrGreater() ? WINDOWS_MODERN : WINDOWS_OLD;
#else
    os_ = LINUX;
#endif
    if (os_ != LINUX) {
        ipc_handle_type_flag_ = hipMemHandleTypeWin32;
    }
    else {
        ipc_handle_type_flag_ = hipMemHandleTypePosixFileDescriptor;
    }

    hipError_t cuda_status = hipGetDevice(&cuda_device_);
}

void CudaInterop::GetDefaultSecurityDescriptor(hipMemAllocationProp* prop) {
    if (os_ == LINUX) {
        return;
    }
    static const char sddl[] = "D:P(OA;;GARCSDWDWOCCDCLCSWLODTWPRPCRFA;;;WD)";
    static OBJECT_ATTRIBUTES obj_attributes;
    static bool obj_attributes_configured = false;

    if (!obj_attributes_configured) {
        PSECURITY_DESCRIPTOR security_descriptor;
        BOOL result = ConvertStringSecurityDescriptorToSecurityDescriptorA(sddl, SDDL_REVISION_1, &security_descriptor, NULL); //NOTE
        if (result == 0) {
            ProgramLog::OutputLine("IPC failure: GetDefaultSecurityDescriptor Failed! (%d)\n", GetLastError());
        }

        InitializeObjectAttributes(&obj_attributes, nullptr, 0, nullptr, security_descriptor); //NOTE

        obj_attributes_configured = true;
    }

    prop->win32HandleMetaData = &obj_attributes;
    return;
}

size_t CudaInterop::RoundWarpGranularity(const size_t& size, const int& granularity) {
    return ((size + granularity - 1) / granularity) * granularity;
}

void CudaInterop::CalculateTotalMemorySize(const size_t& granularity) {
    total_alloc_size_ = 0;

    for (auto& mem_handle : cross_memory_handles_) {
        size_t current_granularity_size = RoundWarpGranularity(mem_handle.size_, granularity);
        total_alloc_size_ += current_granularity_size;
        mem_handle.granularity_size_ = current_granularity_size;
    }
}

void CudaInterop::AddMemoryHandle(const size_t& size) {
    hipMemGenericAllocationHandle_t cuda_position_handle;
    ShareableHandle position_shareable_handle;
    //WIP
    CrossMemoryHandle position_handle = { cuda_position_handle, position_shareable_handle, size };
    cross_memory_handles_.push_back(position_handle);
}

hipError_t CudaInterop::CreateStream(const unsigned int& flags) {
    return hipStreamCreateWithFlags(&cuda_stream_, flags);
}

void CudaInterop::MemoryAllocationProp() {
    current_alloc_prop_ = {};
    current_alloc_prop_.type = hipMemAllocationTypePinned;

    current_alloc_prop_.location.type = hipMemLocationTypeDevice;
    current_alloc_prop_.location.id = cuda_device_;

    current_alloc_prop_.win32HandleMetaData = nullptr;
    current_alloc_prop_.requestedHandleTypes = ipc_handle_type_flag_;
}

void CudaInterop::MemoryAccessDescriptor() {
    access_descriptor_ = {};
    access_descriptor_.location.id = cuda_device_;
    access_descriptor_.location.type = hipMemLocationTypeDevice;
    access_descriptor_.flags = hipMemAccessFlagsProtReadWrite;
}

hipError_t CudaInterop::SimulationSetup() {
    hipDeviceptr_t d_ptr = 0U;
    size_t granularity = 0;

    hipError_t cuda_status = hipSuccess;
    hipError_t cuda_result;

    MemoryAllocationProp();

    GetDefaultSecurityDescriptor(&current_alloc_prop_);

    cuda_result = hipMemGetAllocationGranularity(&granularity, &current_alloc_prop_, hipMemAllocationGranularityRecommended);

    CalculateTotalMemorySize(granularity);

    cuda_result = hipMemAddressReserve(&d_ptr, total_alloc_size_, granularity, 0U, 0);

    cuda_result = hipMemCreate(&cross_memory_handles_[0].cuda_handle_, cross_memory_handles_[0].granularity_size_, &current_alloc_prop_, 0);

    cuda_result = hipMemExportToShareableHandle((void*)&cross_memory_handles_[0].shareable_handle_, cross_memory_handles_[0].cuda_handle_, ipc_handle_type_flag_, 0);

    hipDeviceptr_t va_position = d_ptr; //NOTE: When having other pointers, this will adding the offsets in order to properly account for fitting into the contiguous VA range.
    cross_memory_handles_[0].vulkan_ptr = (void*)va_position;

    cuda_result = hipMemMap(va_position, cross_memory_handles_[0].size_, 0, cross_memory_handles_[0].cuda_handle_, 0);

    cuda_result = hipMemRelease(cross_memory_handles_[0].cuda_handle_);

    MemoryAccessDescriptor();

    hipMemSetAccess(d_ptr, total_alloc_size_, &access_descriptor_, 1); //Adds read-write access to the whole VA range.

    return cuda_status;
}

hipError_t CudaInterop::Clean() {
    hipError_t cuda_result;
    for (const auto& mem_handle : cross_memory_handles_) { //Ensures that all allocations are mapped before attempting unmap memory
        if (!mem_handle.vulkan_ptr) {
            return;
        }
    }

    IPCCloseShareableHandle(cross_memory_handles_[0].shareable_handle_);

    cuda_result = hipMemAddressFree((hipDeviceptr_t) cross_memory_handles_[0].vulkan_ptr, total_alloc_size_);
}

int CudaInterop::IPCCloseShareableHandle(ShareableHandle sh_handle) {
    return CloseHandle(sh_handle);
}