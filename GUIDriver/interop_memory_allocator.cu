#include "interop_memory_allocator.cuh"

InteropMemoryHandler::InteropMemoryHandler() {
	hipError_t cuda_status = hipSuccess;

#ifdef _WIN64
	os_ = IsWindows8OrGreater() ? WINDOWS_MODERN : WINDOWS_OLD;
#else
	os_ = LINUX;
#endif
	if (os_ != LINUX) {
		ipc_handle_type_flag_ = hipMemHandleTypeWin32;
	}
	else {
		ipc_handle_type_flag_ = hipMemHandleTypePosixFileDescriptor;
	}

	cuda_status = hipGetDevice(&cuda_device_); //Inits device

	//Allocates CUDA driver allocation prop settings
	current_alloc_prop_.type = hipMemAllocationTypePinned;
	current_alloc_prop_.location.type = hipMemLocationTypeDevice;
	current_alloc_prop_.location.id = cuda_device_;
	current_alloc_prop_.win32HandleMetaData = nullptr;
	current_alloc_prop_.requestedHandleTypes = ipc_handle_type_flag_;

	//Allocates CUDA driver access descriptor settings
	access_descriptor_ = {};
	access_descriptor_.location.id = cuda_device_;
	access_descriptor_.location.type = hipMemLocationTypeDevice;
	access_descriptor_.flags = hipMemAccessFlagsProtReadWrite;

	if (os_ != LINUX) {
		GetDefaultSecurityDescriptor(&current_alloc_prop_);
	}

	GetAllocationGranularity();
}

InteropMemoryHandler& InteropMemoryHandler::Get() {
	static InteropMemoryHandler instance_;
	return instance_;
}

hipError_t InteropMemoryHandler::GetAllocationGranularity(const hipMemAllocationGranularity_flags flags) {
	hipError_t cuda_result = hipMemGetAllocationGranularity(&granularity, &current_alloc_prop_, flags);
	CudaDriverLog(cuda_result, "Allocation Granularity");

	return cuda_result;
}

void InteropMemoryHandler::GetDefaultSecurityDescriptor(hipMemAllocationProp* prop) {
	if (os_ == LINUX) {
		return;
	}
	static const char sddl[] = "D:P(OA;;GARCSDWDWOCCDCLCSWLODTWPRPCRFA;;;WD)";
	static OBJECT_ATTRIBUTES obj_attributes;
	static bool obj_attributes_configured = false;

	if (!obj_attributes_configured) {
		PSECURITY_DESCRIPTOR security_descriptor;
		BOOL result = ConvertStringSecurityDescriptorToSecurityDescriptorA(sddl, SDDL_REVISION_1, &security_descriptor, NULL); //NOTE
		if (result == 0) {
			ProgramLog::OutputLine("IPC failure: GetDefaultSecurityDescriptor Failed! (%d)\n", GetLastError());
		}

		InitializeObjectAttributes(&obj_attributes, nullptr, 0, nullptr, security_descriptor); //NOTE

		obj_attributes_configured = true;
	}
	prop->win32HandleMetaData = &obj_attributes;
}

size_t InteropMemoryHandler::CalculateTotalMemorySize(const vector<CrossMemoryHandle>& memory_handles, const size_t granularity) {
	size_t total_granularity_size = 0;

	for (auto& mem_handle : cross_memory_handles_) {
		size_t current_granularity_size = RoundWarpGranularity(mem_handle.TotalAllocationSize(), granularity);
		total_granularity_size += current_granularity_size;
		mem_handle.granularity_size = current_granularity_size;
	}

	return total_granularity_size;
}

hipError_t InteropMemoryHandler::CreateNewAllocation() {
	if (allocation_queue_.size() == 0) {
		ProgramLog::OutputLine("Warning: There was no elements in the allocation queue!");
		return hipSuccess;
	}
	//Initialize nullptr equivalent
	size_t local_total_granularity_size = CalculateTotalMemorySize(allocation_queue_, granularity);
	va_ptrs_.push_back(0U);

	//Takes the latest VA CUDA Device pointer and then reserves the total granularity size
	hipError_t cuda_result = hipMemAddressReserve(&va_ptrs_[va_ptrs_.size() - 1], total_granularity_size_, granularity, 0, 0);
	CudaDriverLog(cuda_result, "MemAddressReserve");

	hipDeviceptr_t total = 0U;

	//Traverses through all memory handles
	for (auto& memory_handle : allocation_queue_) {
		//Creates handle, and then exports it
		cuda_result = hipMemCreate(&memory_handle.cuda_handle, memory_handle.granularity_size, &current_alloc_prop_, 0);
		CudaDriverLog(cuda_result, "MemCreate");
		cuda_result = hipMemExportToShareableHandle(&memory_handle.shareable_handle, memory_handle.cuda_handle, ipc_handle_type_flag_, 0);
		CudaDriverLog(cuda_result, "ExportToShareableHandle");

		//Adds stride to memory address
		total += memory_handle.granularity_size;

		//Sets pointer to the handle
		memory_handle.cuda_device_ptr = (void*) memory_handle.cuda_handle;

		//Maps allocation to VA handle
		cuda_result = hipMemMap(total, memory_handle.granularity_size, 0, memory_handle.cuda_handle, 0);
		CudaDriverLog(cuda_result, "MapMemory");

		//Releases memory
		cuda_result = hipMemRelease(memory_handle.cuda_handle);
		CudaDriverLog(cuda_result, "ReleaseMemory");
	}
	//Sets permission for whole VA range
	cuda_result = hipMemSetAccess(va_ptrs_[va_ptrs_.size() - 1], total_granularity_size_, &access_descriptor_, 1);
	CudaDriverLog(cuda_result, "SetMemoryAccess");
	allocation_queue_.clear();

	return cuda_result;
}

hipError_t InteropMemoryHandler::MapExistingPointer(void* ptr, const size_t size, const size_t type_size) {
	if (!ptr) {
		ProgramLog::OutputLine("Warning: This pointer is null!");
	}

	va_ptrs_.push_back((uintptr_t) ptr);

	hipError_t cuda_result = hipMemAddressReserve(&va_ptrs_[va_ptrs_.size() - 1], total_granularity_size_, granularity, 0, 0);
	CudaDriverLog(cuda_result, "MemAddressReserve");

	AddMemoryHandle(size, type_size, false);
	CrossMemoryHandle& current = cross_memory_handles_[cross_memory_handles_.size() - 1];

	cuda_result = hipMemCreate(&current.cuda_handle, current.granularity_size, &current_alloc_prop_, 0);
	CudaDriverLog(cuda_result, "MemCreate");
	cuda_result = hipMemExportToShareableHandle(&current.shareable_handle, current.cuda_handle, ipc_handle_type_flag_, 0);
	CudaDriverLog(cuda_result, "ExportToShareableHandle");

	current.cuda_device_ptr = (void*) current.cuda_handle;

	hipDeviceptr_t va_position = (uintptr_t) va_ptrs_[va_ptrs_.size() - 1];

	cuda_result = hipMemMap(va_position, current.granularity_size, 0, current.cuda_handle, 0);
	CudaDriverLog(cuda_result, "MapMemory");

	cuda_result = hipMemRelease(current.cuda_handle);
	CudaDriverLog(cuda_result, "ReleaseMemory");

	cuda_result = hipMemSetAccess(va_ptrs_[va_ptrs_.size() - 1], total_granularity_size_, &access_descriptor_, 1);
	CudaDriverLog(cuda_result, "SetMemoryAccess");

	return cuda_result;
}

hipError_t InteropMemoryHandler::Clean() {
	hipError_t cuda_result;
	for (const auto& mem_handle : cross_memory_handles_) { //Ensures that all allocations are mapped before attempting to unmap memory
		if (!mem_handle.vulkan_ptr) {
			CudaDriverLog(cuda_result, "Clean");
			return cuda_result;
		}
	}
	for (const auto& mem_handle : cross_memory_handles_) {
		CloseHandle(mem_handle.shareable_handle);

		cuda_result = hipMemAddressFree((hipDeviceptr_t) mem_handle.cuda_device_ptr, total_granularity_size_);
		CudaDriverLog(cuda_result, "VulkanPtrCUDAFree");
	}
	return cuda_result;
}