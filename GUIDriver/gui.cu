﻿
#include "hip/hip_runtime.h"
#include ""

#include "../CUDATest/handler_methods.hpp"

#include "gui_driver.cuh"

#include <stdio.h>

#include <functional>

using std::function;

int WinMain(int argc, char* argv[]) {
    hipError_t cuda_status = hipSuccess;

    function<hipError_t()> set_device_func = []() { return hipSetDevice(0); };
    cuda_status = WrapperFunction(set_device_func, "hipSetDevice failed!", "main",
        cuda_status, "Do you have a CUDA-capable GPU installed?");

    VulkanGUIDriver gui_driver;

    gui_driver.RunGUI();

    cuda_status = hipDeviceReset();
    CudaExceptionHandler(cuda_status, "hipDeviceReset failed!");

    return 0;
}