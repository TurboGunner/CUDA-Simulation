﻿
#include "hip/hip_runtime.h"
#include ""

#include "../CUDATest/handler_methods.hpp"

#include "gui_driver.cuh"

#include <stdio.h>

#include <functional>

//Debugging
#define _CRTDBG_MAP_ALLOC
#include <stdlib.h>
#include <crtdbg.h>

using std::function;

int CALLBACK WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR pCmdLine, int nCmdShow) {
    hipError_t cuda_status = hipSuccess;

     function<hipError_t()> set_device_func = []() { return hipSetDevice(0); };
    cuda_status = WrapperFunction(set_device_func, "hipSetDevice failed!", "main",
        cuda_status, "Do you have a CUDA-capable GPU installed?");

    VulkanGUIDriver gui_driver;

    _CrtDumpMemoryLeaks();
    _CrtSetReportMode(_CRT_WARN, _CRTDBG_MODE_DEBUG);

    gui_driver.RunGUI();

    cuda_status = hipDeviceReset();
    CudaExceptionHandler(cuda_status, "hipDeviceReset failed!");

    return 0;
}