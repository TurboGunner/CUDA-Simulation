﻿
#include "hip/hip_runtime.h"
#include ""

#include "../CUDATest/handler_methods.hpp"

#include "gui_driver.cuh"

#include <stdio.h>

#include <functional>

using std::function;

int CALLBACK WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR pCmdLine, int nCmdShow) {
    hipError_t cuda_status = hipSuccess;

    function<hipError_t()> set_device_func = []() { return hipSetDevice(0); };
    cuda_status = WrapperFunction(set_device_func, "hipSetDevice failed!", "main",
        cuda_status, "Do you have a CUDA-capable GPU installed?");

    VulkanGUIDriver gui_driver;

    gui_driver.RunGUI();

    cuda_status = hipDeviceReset();
    CudaExceptionHandler(cuda_status, "hipDeviceReset failed!");

    return 0;
}