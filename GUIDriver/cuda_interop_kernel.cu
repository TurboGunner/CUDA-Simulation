#include "hip/hip_runtime.h"
#include "cuda_interop_helper.cuh"

__global__ void TestKernel(float* data) {
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	data[x_bounds] = x_bounds;
}

__host__ hipError_t CudaInterop::TestMethod(VkSemaphore& wait_semaphore, VkSemaphore& signal_semaphore) {
	hipError_t cuda_status = hipSuccess;

	void*& device_ptr = cross_memory_handles_[0].cuda_device_ptr,
		*& host_ptr = cross_memory_handles_[0].cuda_host_ptr;
	cuda_status = hipMemsetAsync(device_ptr, 0, cross_memory_handles_[0].TotalAllocationSize(), cuda_stream_);

	dim3 blocks, threads;

	TestKernel<<<blocks, threads, 0, cuda_stream_>>> ((float*) device_ptr);

	cuda_status = PostExecutionChecks(cuda_status, "TestKernel");

	cuda_status = hipMemcpyAsync(host_ptr, device_ptr, cross_memory_handles_[0].TotalAllocationSize(), hipMemcpyDeviceToHost, cuda_stream_);

	return cuda_status;
}

__host__ hipError_t CudaInterop::BulkInitializationTest(VkSemaphore& wait_semaphore, VkSemaphore& signal_semaphore) {
	hipError_t cuda_status = hipSuccess;

	size_t size = 4; //WIP, Test

	AddMemoryHandle(size, sizeof(float)); //Adds memory handle struct

	cuda_status = CreateStream();
	hipError_t cuda_result = SimulationSetupAllocations(); //Setups the allocation for the simulation

	for (auto& cross_memory_handle : cross_memory_handles_) {
		cuda_status = cross_memory_handle.AllocateCudaMemory(); //Allocates CUDA memory across handle structs
	}

	cuda_status = InitializeCudaInterop(wait_semaphore, signal_semaphore);

	return cuda_status;
}

__host__ hipError_t CudaInterop::InteropDrawFrame(VkSemaphore& wait_semaphore, VkSemaphore& signal_semaphore) {
	hipExternalSemaphoreWaitParams wait_params = {};
	wait_params.flags = 0;
	wait_params.params.fence.value = 0;

	hipExternalSemaphoreSignalParams signal_params = {};
	signal_params.flags = 0;
	signal_params.params.fence.value = 0;

	hipError_t cuda_status = hipWaitExternalSemaphoresAsync(&cuda_wait_semaphore_, &wait_params, 1, cuda_stream_);

	cuda_status = TestMethod(wait_semaphore, signal_semaphore);

	hipSignalExternalSemaphoresAsync(&cuda_signal_semaphore_, &signal_params, 1, cuda_stream_);

	return cuda_status;
}

__host__ void CudaInterop::DriverLog(hipError_t& cuda_result, const string& label) {
	const char* name_output, *str_output;

	cuda_result = hipDrvGetErrorName(cuda_result, &name_output);
	cuda_result = hipDrvGetErrorString(cuda_result, &str_output);

	s_stream << "CUDA Driver API Error Status for " << label << ": " << name_output << " | CUDA Driver Error String: " << str_output << std::endl;
	ProgramLog::OutputLine(s_stream);
}