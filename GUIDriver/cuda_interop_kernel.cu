#include "cuda_interop_helper.cuh"

__host__ hipError_t CudaInterop::TestMethod() {
	hipError_t cuda_status = hipSuccess;

	CrossMemoryHandle test = InteropMemoryHandler::CrossMemoryHandles()[0];

	//cuda_status = hipMemsetAsync(test.cuda_device_ptr, 0, test.TotalAllocationSize(), cuda_stream_);
	//CudaExceptionHandler(cuda_status, "CUDAMemsetAsync");

	Grid::SimulateGPU(grid_, cuda_stream_); //WIP
	cuda_status = PostExecutionChecks(cuda_status, "MPMKernel");

	float* host_test = new float();

	cuda_status = hipMemcpyAsync(host_test, &((Vector3D*)test.cuda_device_ptr)[163].dim[0], sizeof(float), hipMemcpyDeviceToHost, cuda_stream_);

	//hipFree(cross_memory_handles_[0].cuda_device_ptr);

	s_stream << "Host Test: " << *host_test << std::endl; //WIP, DEBUG!
	ProgramLog::OutputLine(s_stream); //WIP, DEBUG!

	delete host_test;

	return cuda_status;
}

__host__ hipError_t CudaInterop::BulkInitializationTest(VkSemaphore& wait_semaphore, VkSemaphore& signal_semaphore, const size_t& size) {
	hipError_t cuda_status = hipSuccess;

	cuda_status = CreateStream();
	ProgramLog::OutputLine("Creating CUDA async stream!\n");

	//hipError_t cuda_result = InteropMemoryHandler::MapExistingPointer(grid_->particle_position_device_, size, sizeof(Vector3D));
	InteropMemoryHandler::AddMemoryHandle(grid_->GetParticleCount(), sizeof(Vector3D), true);
	hipError_t cuda_result = InteropMemoryHandler::CreateNewAllocation();
	grid_->particle_position_device_ = (Vector3D*) InteropMemoryHandler::CrossMemoryHandles()[0].cuda_device_ptr;

	//hipError_t cuda_result = SimulationSetupAllocations(); //Setups the allocation for the simulation
	ProgramLog::OutputLine("Setting up simulation interop allocations!");

	cuda_status = InitializeCudaInterop(wait_semaphore, signal_semaphore);

	CudaExceptionHandler(cuda_status, "CUDA Interop");

	return cuda_status;
}

__host__ hipError_t CudaInterop::InteropDrawFrame(VkSemaphore& wait_semaphore, VkSemaphore& signal_semaphore) {
	hipExternalSemaphoreWaitParams wait_params = {};
	wait_params.flags = 0;
	wait_params.params.fence.value = 0;

	hipExternalSemaphoreSignalParams signal_params = {};
	signal_params.flags = 0;
	signal_params.params.fence.value = 0;

	hipError_t cuda_status = hipWaitExternalSemaphoresAsync(&cuda_wait_semaphore_, &wait_params, 1, cuda_stream_);
	CudaExceptionHandler(cuda_status, "CUDAWaitExternalSemaphoreAsync");

	cuda_status = TestMethod();
	CudaExceptionHandler(cuda_status, "ExecuteMethod");

	cuda_status = hipSignalExternalSemaphoresAsync(&cuda_signal_semaphore_, &signal_params, 1, cuda_stream_);
	CudaExceptionHandler(cuda_status, "CUDASignalExternalSemaphoreAsync");

	return cuda_status;
}