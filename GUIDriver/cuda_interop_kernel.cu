#include "cuda_interop_helper.cuh"

__host__ hipError_t CudaInterop::TestMethod() {
	hipError_t cuda_status = hipSuccess;

	cuda_status = hipMemsetAsync(cross_memory_handles_[0].cuda_device_ptr, 0, cross_memory_handles_[0].TotalAllocationSize(), cuda_stream_);
	CudaExceptionHandler(cuda_status, "CUDAMemsetAsync");

	//dim3 blocks, threads;

	Grid::SimulateGPU(grid_, cuda_stream_); //WIP

	cuda_status = PostExecutionChecks(cuda_status, "MPMKernel");

	//hipStreamSynchronize(cuda_stream_);

	float* host_test = new float();

	cuda_status = hipMemcpyAsync(host_test, &((Vector3D*) cross_memory_handles_[0].cuda_device_ptr)[163].dim[0], sizeof(float), hipMemcpyDeviceToHost, cuda_stream_);

	//hipFree(cross_memory_handles_[0].cuda_device_ptr);

	s_stream << "Host Test: " << *host_test << std::endl; //WIP, DEBUG!
	ProgramLog::OutputLine(s_stream); //WIP, DEBUG!

	s_stream << "Pointer Test: " << cross_memory_handles_[0].vulkan_ptr << std::endl; //WIP, DEBUG!
	ProgramLog::OutputLine(s_stream); //WIP, DEBUG!

	delete host_test;
	//cuda_status = hipMemcpyAsync(host_ptr, device_ptr, cross_memory_handles_[0].TotalAllocationSize(), hipMemcpyDeviceToHost, cuda_stream_); //NOTE: TEST

	return cuda_status;
}

__host__ hipError_t CudaInterop::BulkInitializationTest(VkSemaphore& wait_semaphore, VkSemaphore& signal_semaphore, const size_t& size) {
	hipError_t cuda_status = hipSuccess;

	AddMemoryHandle(size, sizeof(Vector3D)); //Adds memory handle struct

	cuda_status = CreateStream();
	ProgramLog::OutputLine("Creating CUDA async stream!\n");

	cross_memory_handles_[0].cuda_device_ptr = grid_->particle_position_device_;

	hipError_t cuda_result = SimulationSetupAllocations(); //Setups the allocation for the simulation
	ProgramLog::OutputLine("Setting up simulation interop allocations!");

	//for (auto& cross_memory_handle : cross_memory_handles_) {
		//cuda_status = cross_memory_handle.AllocateCudaMemory(); //Allocates CUDA memory across handle structs
	//}
	ProgramLog::OutputLine("CUDA memory allocated successfully!");

	cuda_status = InitializeCudaInterop(wait_semaphore, signal_semaphore);

	CudaExceptionHandler(cuda_status, "CUDA Interop");

	return cuda_status;
}

__host__ hipError_t CudaInterop::InteropDrawFrame(VkSemaphore& wait_semaphore, VkSemaphore& signal_semaphore) {
	hipExternalSemaphoreWaitParams wait_params = {};
	wait_params.flags = 0;
	wait_params.params.fence.value = 0;

	hipExternalSemaphoreSignalParams signal_params = {};
	signal_params.flags = 0;
	signal_params.params.fence.value = 0;

	hipError_t cuda_status = hipWaitExternalSemaphoresAsync(&cuda_wait_semaphore_, &wait_params, 1, cuda_stream_);
	CudaExceptionHandler(cuda_status, "CUDAWaitExternalSemaphoreAsync");

	cuda_status = TestMethod();
	CudaExceptionHandler(cuda_status, "ExecuteMethod");

	cuda_status = hipSignalExternalSemaphoresAsync(&cuda_signal_semaphore_, &signal_params, 1, cuda_stream_);
	CudaExceptionHandler(cuda_status, "CUDASignalExternalSemaphoreAsync");

	return cuda_status;
}