#include "hip/hip_runtime.h"
#include "cuda_interop_helper.cuh"

__global__ void TestKernel(float* data) {
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	data[x_bounds] = x_bounds;
}

__host__ hipError_t CudaInterop::TestMethod(VkSemaphore& wait_semaphore, VkSemaphore& signal_semaphore) {
	hipError_t cuda_status = hipSuccess;

	void*& device_ptr = cross_memory_handles_[0].cuda_device_ptr,
		*& host_ptr = cross_memory_handles_[0].cuda_host_ptr;
	cuda_status = hipMemsetAsync(device_ptr, 0, cross_memory_handles_[0].TotalAllocationSize(), cuda_stream_);
	CudaExceptionHandler(cuda_status, "CUDAMemsetAsync");

	//dim3 blocks, threads;

	//TestKernel<<<blocks, threads, 0, cuda_stream_>>> ((float*) device_ptr);
	Grid::SimulateGPU(grid_, cuda_stream_); //WIP

	cuda_status = PostExecutionChecks(cuda_status, "MPMKernel");

	//cuda_status = hipMemcpyAsync(host_ptr, device_ptr, cross_memory_handles_[0].TotalAllocationSize(), hipMemcpyDeviceToHost, cuda_stream_); //NOTE: TEST

	return cuda_status;
}

__host__ hipError_t CudaInterop::BulkInitializationTest(VkSemaphore& wait_semaphore, VkSemaphore& signal_semaphore, const size_t& size) {
	hipError_t cuda_status = hipSuccess;

	AddMemoryHandle(size, sizeof(Vector3D)); //Adds memory handle struct

	cuda_status = CreateStream();
	ProgramLog::OutputLine("Creating CUDA async stream!");

	hipError_t cuda_result = SimulationSetupAllocations(); //Setups the allocation for the simulation
	ProgramLog::OutputLine("Setting up simulation interop allocations!");

	//for (auto& cross_memory_handle : cross_memory_handles_) {
		//cuda_status = cross_memory_handle.AllocateCudaMemory(); //Allocates CUDA memory across handle structs
	//}
	ProgramLog::OutputLine("CUDA memory allocated successfully!");

	cross_memory_handles_[0].cuda_device_ptr = grid_->particle_velocity_device_;

	cuda_status = InitializeCudaInterop(wait_semaphore, signal_semaphore);

	return cuda_status;
}

__host__ hipError_t CudaInterop::InteropDrawFrame(VkSemaphore& wait_semaphore, VkSemaphore& signal_semaphore) {
	hipExternalSemaphoreWaitParams wait_params = {};
	wait_params.flags = 0;
	wait_params.params.fence.value = 0;

	hipExternalSemaphoreSignalParams signal_params = {};
	signal_params.flags = 0;
	signal_params.params.fence.value = 0;

	hipError_t cuda_status = hipWaitExternalSemaphoresAsync(&cuda_wait_semaphore_, &wait_params, 1, cuda_stream_);
	CudaExceptionHandler(cuda_status, "CUDAWaitExternalSemaphoreAsync");

	cuda_status = TestMethod(wait_semaphore, signal_semaphore);
	CudaExceptionHandler(cuda_status, "ExecuteMethod");

	cuda_status = hipSignalExternalSemaphoresAsync(&cuda_signal_semaphore_, &signal_params, 1, cuda_stream_);
	CudaExceptionHandler(cuda_status, "CUDASignalExternalSemaphoreAsync");

	return cuda_status;
}

__host__ void CudaInterop::DriverLog(hipError_t& cuda_result, const string& label) {
	const char* name_output, *str_output;

	cuda_result = hipDrvGetErrorName(cuda_result, &name_output);
	cuda_result = hipDrvGetErrorString(cuda_result, &str_output);

	s_stream << "CUDA Driver API Error Status for " << label << ": " << name_output << " | CUDA Driver Error String: " << str_output << std::endl;
	ProgramLog::OutputLine(s_stream);
}