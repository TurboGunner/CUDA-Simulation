#include "hip/hip_runtime.h"
#include "matrix.cuh"

__host__ __device__ Matrix::Matrix(const size_t& rows_in, const size_t& columns_in, const bool& local_in) {
    rows = rows_in;
    columns = columns_in;

    hipError_t cuda_status = hipSuccess;

    is_square = rows == columns;

    size_t size_alloc = rows * columns * sizeof(float);

    local = local_in;

#ifdef __CUDA_ARCH__
    data_device = (float*) malloc(size_alloc);
#else
    cuda_status = hipHostMalloc(&data, size_alloc);
    CudaExceptionHandler(cuda_status, "Could not allocate the memory for the matrix (host).");
    if (!local) {
        cuda_status = hipMalloc(&data_device, size_alloc);
        CudaExceptionHandler(cuda_status, "Could not allocate the memory for the matrix (device, on host).");
    }
#endif
}


__host__ __device__ Matrix* Matrix::Create(const size_t& rows, const size_t& columns, const bool& local) {
    Matrix* matrix = nullptr;
    hipError_t cuda_status = hipSuccess;

#ifdef __CUDA_ARCH__
    cuda_status = hipMalloc(&matrix->device_alloc, sizeof(Matrix));
    if (cuda_status != hipSuccess) {
        printf("%s", "Error: Did not properly allocate matrix pointer (device, on device).");
    }
    *matrix = Matrix(rows, columns, local);
    *(matrix->device_alloc) = Matrix(rows, columns, local);
#else
    cuda_status = hipHostMalloc(&matrix, sizeof(Matrix));
    CudaExceptionHandler(cuda_status, "Could not allocate the memory for the matrix pointer (host).");
    *matrix = Matrix(rows, columns, local);
    if (!local) {
        cuda_status = hipMalloc(&(matrix->device_alloc), sizeof(Matrix));
        CudaExceptionHandler(cuda_status, "Could not allocate the memory for the matrix pointer (device, on host).");
    }
#endif
    return matrix;
}

__host__ __device__ size_t Matrix::IX(size_t row, size_t column) const {
    return column + (rows * row);
}

__host__ __device__ float& Matrix::Get(const int& index) {
    if (index >= rows * columns || index < 0) {
        printf("%s%d\n", "Warning: Out of bounds! Index: ", index);
#ifdef __CUDA_ARCH__
        return data_device[0];
#else
        return data[0];
#endif
    }
#ifdef __CUDA_ARCH__
    return data_device[index];
#else
    return data[index];
#endif
}

__host__ __device__ float& Matrix::Get(const size_t& row, const size_t& column) {
    return Get(IX(row, column));
}

__host__ __device__ float& Matrix::operator[](const int& index) {
    return Get(index);
}

__host__ __device__ void Matrix::Set(const float& value, const int& index) {
    if (index >= rows * columns || index < 0) {
        printf("%s\n", "Warning: Out of bounds!");
        return;
    }
#ifdef __CUDA_ARCH__
    data_device[index] = value;
#else
    data[index] = value;
#endif
}

__host__ __device__ void Matrix::Set(const float& value, const size_t& row, const size_t& column) {
    Set(value, IX(row, column));
}

__host__ hipError_t Matrix::HostTransfer() {
    hipError_t cuda_status = hipSuccess;
    cuda_status = CopyFunction("HostTransferTable", data, data_device, hipMemcpyDeviceToHost, cuda_status, sizeof(float), rows * columns);
    hipDeviceSynchronize();

    return cuda_status;
}

__host__ hipError_t Matrix::DeviceTransfer(Matrix* ptr, Matrix* src) {
    hipError_t cuda_status = hipSuccess;

    if (!device_allocated_status) {
        cuda_status = hipMalloc(&ptr, sizeof(Matrix));
        device_allocated_status = true;
        cuda_status = CopyFunction("DeviceTransferObject", ptr, src, hipMemcpyHostToDevice, cuda_status, sizeof(Matrix), 1);
        device_alloc = ptr;
        if (local) {
            std::cout << "Warning! Host locality is set to true, but device synchronization was called." <<
                "\n\nThis will likely result in a segfault, as the corresponding GPU table data was not initialized." << std::endl;
        }
    }
    else {
        ptr = device_alloc;
    }

    cuda_status = CopyFunction("DeviceTransferTable", data_device, data, hipMemcpyHostToDevice, cuda_status, sizeof(float), rows * columns);
    hipDeviceSynchronize();

    return cuda_status;
}

__host__ __device__ void Matrix::PrintMatrix(const char* label) {
    if (label) { // Nullptr default check
        printf("\n\n%s", label);
    }
    else {
        printf("\n\n");
    }
    for (size_t i = 0; i < rows; i++) {
        printf("\n");
        for (size_t j = 0; j < columns; j++) {
            printf("%f ", Get(IX(j, i)));
        }
    }
}

__host__ __device__ float* Matrix::Row(const size_t& index) {
    float* output = (float*) malloc(columns * sizeof(float));
    for (int i = 0; i < columns; i++) {
        output[i] = Get(i, index);
    }
    return output;
}

__host__ __device__ float* Matrix::Column(const size_t& index) {
    float* output = (float*)malloc(rows * sizeof(float));
    for (int i = 0; i < rows; i++) {
        output[i] = Get(index, i);
    }
    return output;
}

__host__ __device__ hipError_t Matrix::Destroy() {
    hipError_t cuda_status = hipSuccess;
    if (!local) {
        cuda_status = hipFree(data_device);
        if (cuda_status != hipSuccess) {
            printf("%s\n", "Could not free memory for the data device.");
        }
    }
    if (device_allocated_status) {
        cuda_status = hipFree(device_alloc);
        printf("%s\n", "Could not free memory for the device allocation.");
    }
#ifndef __CUDA_ARCH__
    free(data);
#endif
    return cuda_status;
}

__host__ void Matrix::DeleteAllocations(vector<Matrix*> matrices) {
    for (auto matrix : matrices) {
        matrix->Destroy();
    }
}

hipError_t Matrix::PopulateRandomHost(Matrix* matrix, const float& min, const float& max) {
    RandomFloat random(min, max, 3);

    for (size_t i = 0; i < matrix->rows; i++) {
        for (size_t j = 0; j < matrix->columns; j++) {
            matrix->Set(random.Generate(), j, i);
        }
    }

    return matrix->DeviceTransfer(matrix->device_alloc, matrix);
}