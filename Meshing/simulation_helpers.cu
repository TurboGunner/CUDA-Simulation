#include "hip/hip_runtime.h"
#include "mpm.cuh"

inline void CurandCallCheck(const hiprandStatus_t& status) {
	if (status != HIPRAND_STATUS_SUCCESS) {
		ProgramLog::OutputLine("Warning! CUDA Rand Status is not successful! State: " + std::to_string(status));
	}
}

__host__ void Grid::CalculateBounds() {
	unsigned int threads_per_dim = 8; //512 in total per block, max as each block can have 1024 threads only
	unsigned int block_count = ((side_size_ + threads_per_dim) - 1) / (threads_per_dim);

	cell_threads = dim3(threads_per_dim, threads_per_dim, threads_per_dim);
	cell_blocks = dim3(block_count, block_count, block_count);

	ProgramLog::OutputLine("Resolution: " + std::to_string(GetResolution()));

	//Resolution is now on blocks due to resolution being problematic with the thread count limits for blocks
	particle_threads = dim3(threads_per_dim, threads_per_dim, threads_per_dim);
	particle_blocks = dim3(block_count * GetResolution(), block_count, block_count);

	size_t thread_count = pow(block_count, 3) * pow(threads_per_dim, 3);

	ProgramLog::OutputLine("Threads (Total): " + std::to_string(thread_count));

	ProgramLog::OutputLine("Cells (Total): " + std::to_string(total_size_));
}

__host__ void GenerateRandomParticles(Grid* grid) {
	hiprandGenerator_t gen;
	srand(time(nullptr));

	int seed = rand();

	hiprandStatus_t curand_status;

	curand_status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	CurandCallCheck(curand_status);
	curand_status = hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	CurandCallCheck(curand_status);
	curand_status = hiprandGenerateUniform(gen, grid->particle_mass_device_, grid->GetParticleCount());
	CurandCallCheck(curand_status);
	curand_status = hiprandDestroyGenerator(gen);
	CurandCallCheck(curand_status);
}

__host__ static hipError_t DebugGPU(Grid* grid, hipStream_t& hip_stream) {
	Vector3D* host_test = new Vector3D();
	hipError_t cuda_status = hipStreamSynchronize(hip_stream); //WIP, DEBUG!

	cuda_status = hipMemcpy(host_test, &grid->particle_position_device_[16366], sizeof(Vector3D), hipMemcpyDeviceToHost); //WIP, DEBUG!

	s_stream << host_test->x() << " " << host_test->y() << " " << host_test->z() << std::endl; //WIP, DEBUG!
	ProgramLog::OutputLine(s_stream); //WIP, DEBUG!
	delete host_test;

	return cuda_status;
}

__host__ hipError_t Grid::SimulateGPU(Grid* grid, hipStream_t& hip_stream) {
	hipError_t cuda_status = hipSuccess;

	dim3& blocks = grid->cell_blocks, &threads = grid->cell_threads;
	dim3& blocks2 = grid->particle_blocks, &threads2 = grid->particle_threads;

	if (!grid->up_to_date_) {
		grid->CalculateBounds();
		grid->DeviceTransfer(grid);

		GenerateRandomParticles(grid);

		grid->up_to_date_ = true;
		cuda_status = hipStreamSynchronize(hip_stream);
	}

	ClearGrid<<<blocks, threads, 0, hip_stream>>> (grid);
	cuda_status = PostExecutionChecks(cuda_status, "ClearGrid", false);
	CudaExceptionHandler(cuda_status, "ClearGrid failed!");

	DebugGPU(grid, hip_stream);

	UpdateCell<<<blocks2, threads2, 0, hip_stream>>> (grid, grid->momentum_matrix->device_alloc, grid->cell_dist_matrix->device_alloc, grid->momentum->device_alloc);
	cuda_status = PostExecutionChecks(cuda_status, "CellMomentum", false);
	CudaExceptionHandler(cuda_status, "CellMomentum failed!");

	DebugGPU(grid, hip_stream);

	SimulateGrid<<<blocks2, threads2, 0, hip_stream>>> (grid, grid->stress_matrix->device_alloc, grid->momentum->device_alloc, grid->viscosity_term->device_alloc); //NOTE
	cuda_status = PostExecutionChecks(cuda_status, "VelocityGradientSolve", false);
	CudaExceptionHandler(cuda_status, "SimulateGrid failed!");

	DebugGPU(grid, hip_stream);

	UpdateGrid<<<blocks, threads, 0, hip_stream>>> (grid);
	cuda_status = PostExecutionChecks(cuda_status, "UpdateGrid", false);
	CudaExceptionHandler(cuda_status, "UpdateGrid failed!");

	DebugGPU(grid, hip_stream);

	AdvectParticles<<<blocks2, threads2, 0, hip_stream>>> (grid, grid->B_term->device_alloc, grid->weighted_term->device_alloc);
	cuda_status = PostExecutionChecks(cuda_status, "AdvectParticles", false);
	CudaExceptionHandler(cuda_status, "AdvectParticles failed!");

	DebugGPU(grid, hip_stream);

	return cuda_status;
}