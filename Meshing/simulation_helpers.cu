#include "hip/hip_runtime.h"
#include "mpm.cuh"

inline void CurandCallCheck(const hiprandStatus_t& status) {
	if (status != HIPRAND_STATUS_SUCCESS) {
		ProgramLog::OutputLine("Warning! CUDA Rand Status is not successful! State: " + std::to_string(status));
	}
}

__global__ void UpdateGrid(Grid* grid) {
	//Cell Boundaries
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z_bounds = blockIdx.z * blockDim.z + threadIdx.z;

	IndexPair incident(x_bounds, y_bounds, z_bounds); //Current Index
	if (grid->GetCellMass(incident) > 0) {
		grid->GetCellVelocity(incident) /= grid->GetCellMass(incident); //Converting momentum to velocity
		printf("%f\n", grid->GetCellMass(incident));

		//Applying gravity to velocity
		Vector3D gravity_vector(0.0f, 0.0f, grid->gravity);
		grid->GetCellVelocity(incident) += (gravity_vector * grid->dt);

		//Boundary Conditions
		if (x_bounds < 2 || x_bounds > grid->side_size_ - 3) {
			grid->GetCellVelocity(incident).dim[0] = 0.0f;
		}
		if (y_bounds < 2 || y_bounds > grid->side_size_ - 3) {
			grid->GetCellVelocity(incident).dim[1] = 0.0f;
		}
		if (z_bounds < 2 || z_bounds > grid->side_size_ - 3) {
			grid->GetCellVelocity(incident).dim[2] = 0.0f;
		}
	}
}

__global__ void ClearGrid(Grid* grid) {
	//Cell Boundaries
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z_bounds = blockIdx.z * blockDim.z + threadIdx.z;

	IndexPair incident(x_bounds, y_bounds, z_bounds); //Current position

	grid->GetCellMass(incident) = 0.0f;
	grid->GetCellVelocity(incident).Reset();
}

__global__ void SetValue(Grid* grid) {
	//Cell Boundaries
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z_bounds = blockIdx.z * blockDim.z + threadIdx.z;

	IndexPair incident(x_bounds, y_bounds, z_bounds); //Current position

	Vector3D vector_pos((float) x_bounds / grid->GetResolution(), y_bounds, z_bounds);

	for (int i = 0; i < 3; i++) { //NOTE
		if (vector_pos.dim[i] > grid->side_size_ - 2) {
			vector_pos.dim[i] -= 2;
		}
		if (vector_pos.dim[i] < 2) {
			vector_pos.dim[i] += 2;
		}
	}

	grid->GetPosition(incident) = vector_pos;
	assert(vector_pos.dim[0] <= grid->side_size_ && vector_pos.dim[0] >= 0.0f);
	assert(vector_pos.dim[1] <= grid->side_size_ && vector_pos.dim[1] >= 0.0f);
	assert(vector_pos.dim[2] <= grid->side_size_ && vector_pos.dim[2] >= 0.0f);
}

__host__ void Grid::CalculateBounds() {
	unsigned int threads_per_dim = 8; //512 in total per block, max as each block can have 1024 threads only
	unsigned int block_count = ((side_size_ + threads_per_dim) - 1) / (threads_per_dim);

	cell_threads = dim3(threads_per_dim, threads_per_dim, threads_per_dim);
	cell_blocks = dim3(block_count, block_count, block_count);

	ProgramLog::OutputLine("Resolution: " + std::to_string(GetResolution()));

	//Resolution is now on blocks due to resolution being problematic with the thread count limits for blocks
	particle_threads = dim3(threads_per_dim, threads_per_dim, threads_per_dim);
	particle_blocks = dim3(block_count * GetResolution(), block_count, block_count);

	size_t thread_count = pow(block_count, 3) * pow(threads_per_dim, 3);

	ProgramLog::OutputLine("Threads (Total): " + std::to_string(thread_count));

	ProgramLog::OutputLine("Cells (Total): " + std::to_string(total_size_));
}

__host__ void GenerateRandomParticles(Grid* grid) {
	hiprandGenerator_t gen;
	srand(time(nullptr));

	int seed = rand();

	hiprandStatus_t curand_status;

	curand_status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	CurandCallCheck(curand_status);
	curand_status = hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	CurandCallCheck(curand_status);
	curand_status = hiprandGenerateUniform(gen, grid->particle_mass_device_, grid->GetParticleCount());
	//curand_status = hiprandGenerateUniform(gen, (float*) grid->particle_position_device_, grid->GetParticleCount() * (sizeof(Vector3D) / sizeof(float)));
	CurandCallCheck(curand_status);
	curand_status = hiprandDestroyGenerator(gen);
	CurandCallCheck(curand_status);
	//Vector3D vec_test(1.0f, 1.0f, 1.0f);

	//hipMemcpy(&grid->particle_position_device_[163], &vec_test, sizeof(Vector3D), hipMemcpyHostToDevice);
}

__host__ static hipError_t DebugGPU(Grid* grid, hipStream_t& hip_stream) {
	Vector3D* host_test = new Vector3D();
	hipError_t cuda_status = hipStreamSynchronize(hip_stream); //WIP, DEBUG!

	cuda_status = hipMemcpy(host_test, &grid->particle_position_device_[15563], sizeof(Vector3D), hipMemcpyDeviceToHost); //WIP, DEBUG!

	s_stream << host_test->x() << " " << host_test->y() << " " << host_test->z() << std::endl; //WIP, DEBUG!
	ProgramLog::OutputLine(s_stream); //WIP, DEBUG!
	delete host_test;

	return cuda_status;
}

//Maybe make weights a shared memory allocation? Maybe a static shared memory context that is accessible through all the sim kernels?
__device__ Vector3D* GetWeights(Vector3D cell_difference) { //Returns weights shared
	Vector3D weights[3] {}; //Array of weights

	weights[0] = (cell_difference.Negative() + 0.5f).Squared() * 0.5f;
	weights[1] = cell_difference.Squared().Negative() + 0.75f;
	weights[2] = (cell_difference + 0.5f).Squared() * 0.5f;

	return weights;
}

__host__ hipError_t Grid::SimulateGPU(Grid* grid, hipStream_t& hip_stream) {
	hipError_t cuda_status = hipSuccess;

	dim3& blocks = grid->cell_blocks, &threads = grid->cell_threads;
	dim3& blocks2 = grid->particle_blocks, &threads2 = grid->particle_threads;

	if (!grid->up_to_date_) {
		grid->CalculateBounds();
		grid->DeviceTransfer(grid);

		GenerateRandomParticles(grid);

		SetValue<<<blocks2, threads2, 0, hip_stream>>> (grid);
		cuda_status = hipStreamSynchronize(hip_stream);

		grid->up_to_date_ = true;
	}

	ProgramLog::OutputLine("\n\nBREAK\n\n");

	ClearGrid<<<blocks, threads, 0, hip_stream>>> (grid);

	cuda_status = PostExecutionChecks(cuda_status, "ClearGrid", false);
	CudaExceptionHandler(cuda_status, "ClearGrid failed!");

	DebugGPU(grid, hip_stream);

	UpdateCell<<<blocks2, threads2, 0, hip_stream>>> (grid);
	cuda_status = PostExecutionChecks(cuda_status, "CellMomentum", false);
	CudaExceptionHandler(cuda_status, "CellMomentum failed!");

	DebugGPU(grid, hip_stream);

	SimulateGrid<<<blocks2, threads2, 0, hip_stream>>> (grid); //NOTE
	cuda_status = PostExecutionChecks(cuda_status, "VelocityGradientSolve", false);
	CudaExceptionHandler(cuda_status, "SimulateGrid failed!");

	DebugGPU(grid, hip_stream);

	UpdateGrid<<<blocks, threads, 0, hip_stream>>> (grid);
	cuda_status = PostExecutionChecks(cuda_status, "UpdateGrid", false);
	CudaExceptionHandler(cuda_status, "UpdateGrid failed!");

	DebugGPU(grid, hip_stream);

	AdvectParticles<<<blocks2, threads2, 0, hip_stream>>> (grid);
	cuda_status = PostExecutionChecks(cuda_status, "AdvectParticles", false);
	CudaExceptionHandler(cuda_status, "AdvectParticles failed!");

	hipStreamSynchronize(hip_stream);

	DebugGPU(grid, hip_stream);

	return cuda_status;
}