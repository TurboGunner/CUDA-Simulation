#include "hip/hip_runtime.h"
#include "mpm.cuh"

inline void CurandCallCheck(const hiprandStatus_t& status) {
	if (status != HIPRAND_STATUS_SUCCESS) {
		ProgramLog::OutputLine("Warning! CUDA Rand Status is not successful! State: " + std::to_string(status));
	}
}

__global__ void UpdateGrid(Grid* grid) {
	//Cell Boundaries
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z_bounds = blockIdx.z * blockDim.z + threadIdx.z;

	IndexPair incident(x_bounds, y_bounds, z_bounds); //Current Index

	grid->GetCellVelocity(incident) /= grid->GetCellMass(incident); //Converting momentum to velocity

	//Applying gravity to velocity
	Vector3D gravity_vector(0.0f, 0.0f, grid->gravity);
	grid->GetCellVelocity(incident) += (gravity_vector * grid->dt);

	//Boundary Conditions
	if (x_bounds < 2 || x_bounds > grid->side_size_ - 3) {
		grid->GetCellVelocity(incident).dim[0] = 0.0f;
	}
	if (y_bounds < 2 || y_bounds > grid->side_size_ - 3) {
		grid->GetCellVelocity(incident).dim[1] = 0.0f;
	}
	if (z_bounds < 2 || z_bounds > grid->side_size_ - 3) {
		grid->GetCellVelocity(incident).dim[2] = 0.0f;
	}
}

__global__ void ClearGrid(Grid* grid) {
	//Cell Boundaries
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z_bounds = blockIdx.z * blockDim.z + threadIdx.z;

	IndexPair incident(x_bounds, y_bounds, z_bounds); //Current position

	grid->GetCellMass(incident) = 0;
	grid->GetCellVelocity(incident).Reset();
}

__host__ void Grid::CalculateBounds() {
	unsigned int threads_per_dim = 8; //512 in total per block, max as each block can have 1024 threads only
	unsigned int block_count = ((side_size_ + threads_per_dim) - 1) / (threads_per_dim);

	cell_threads = dim3(threads_per_dim, threads_per_dim, threads_per_dim);
	cell_blocks = dim3(block_count, block_count, block_count);

	ProgramLog::OutputLine("Resolution: " + std::to_string(GetResolution()));

	//Resolution is now on blocks due to resolution being problematic with the thread count limits for blocks
	particle_threads = dim3(threads_per_dim, threads_per_dim, threads_per_dim);
	particle_blocks = dim3(block_count * GetResolution(), block_count, block_count);

	size_t thread_count = pow(block_count, 3) * pow(threads_per_dim, 3);

	ProgramLog::OutputLine("Threads (Total): " + std::to_string(thread_count));

	ProgramLog::OutputLine("Cells (Total): " + std::to_string(total_size_));
}

__host__ void GenerateRandomParticles(Grid* grid) {
	hiprandGenerator_t gen;
	srand(time(nullptr));

	int seed = rand();

	hiprandStatus_t curand_status;

	curand_status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	CurandCallCheck(curand_status);
	curand_status = hiprandSetPseudoRandomGeneratorSeed(gen, seed);
	CurandCallCheck(curand_status);
	curand_status = hiprandGenerateUniform(gen, grid->particle_mass_device_, grid->GetParticleCount());
	//curand_status = hiprandGenerateUniform(gen, (float*) grid->particle_velocity_device_, grid->GetParticleCount() * (sizeof(Vector3D) / sizeof(float)));
	CurandCallCheck(curand_status);
	curand_status = hiprandDestroyGenerator(gen);
	CurandCallCheck(curand_status);
	Vector3D vec_test(1.0f, 1.0f, 1.0f);

	hipMemcpy(&grid->particle_position_device_[163], &vec_test, sizeof(Vector3D), hipMemcpyHostToDevice);
}

__host__ static hipError_t DebugGPU(Grid* grid, hipStream_t& hip_stream) {
	Vector3D* host_test = new Vector3D();
	hipError_t cuda_status = hipStreamSynchronize(hip_stream); //WIP, DEBUG!

	cuda_status = hipMemcpy(host_test, &grid->particle_position_device_[16366], sizeof(Vector3D), hipMemcpyDeviceToHost); //WIP, DEBUG!

	s_stream << host_test->x() << " " << host_test->y() << " " << host_test->z() << std::endl; //WIP, DEBUG!
	ProgramLog::OutputLine(s_stream); //WIP, DEBUG!
	delete host_test;

	return cuda_status;
}

//Maybe make weights a shared memory allocation? Maybe a static shared memory context that is accessible through all the sim kernels?
__device__ Vector3D* GetWeights(Vector3D cell_difference) { //Returns weights shared
	Vector3D weights[3] {}; //Array of weights

	weights[0] = (cell_difference.Negative() + 0.5f).Squared() * 0.5f;
	weights[1] = cell_difference.Squared().Negative() + 0.75f;
	weights[2] = (cell_difference + 0.5f).Squared() * 0.5f;

	return weights;
}

__host__ hipError_t Grid::SimulateGPU(Grid* grid, hipStream_t& hip_stream) {
	hipError_t cuda_status = hipSuccess;

	dim3& blocks = grid->cell_blocks, &threads = grid->cell_threads;
	dim3& blocks2 = grid->particle_blocks, &threads2 = grid->particle_threads;

	if (!grid->up_to_date_) {
		grid->CalculateBounds();
		grid->DeviceTransfer(grid);

		GenerateRandomParticles(grid);

		grid->up_to_date_ = true;
		//cuda_status = hipStreamSynchronize(hip_stream);
	}

	ClearGrid<<<blocks, threads, 0, hip_stream>>> (grid);
	cuda_status = PostExecutionChecks(cuda_status, "ClearGrid", false);
	CudaExceptionHandler(cuda_status, "ClearGrid failed!");

	//DebugGPU(grid, hip_stream);

	UpdateCell<<<blocks2, threads2, 0, hip_stream>>> (grid, grid->momentum_matrix->device_alloc, grid->cell_dist_matrix->device_alloc, grid->momentum->device_alloc);
	cuda_status = PostExecutionChecks(cuda_status, "CellMomentum", false);
	CudaExceptionHandler(cuda_status, "CellMomentum failed!");

	//DebugGPU(grid, hip_stream);

	SimulateGrid<<<blocks2, threads2, 0, hip_stream>>> (grid, grid->stress_matrix->device_alloc, grid->momentum->device_alloc, grid->viscosity_term->device_alloc); //NOTE
	cuda_status = PostExecutionChecks(cuda_status, "VelocityGradientSolve", false);
	CudaExceptionHandler(cuda_status, "SimulateGrid failed!");

	//DebugGPU(grid, hip_stream);

	UpdateGrid<<<blocks, threads, 0, hip_stream>>> (grid);
	cuda_status = PostExecutionChecks(cuda_status, "UpdateGrid", false);
	CudaExceptionHandler(cuda_status, "UpdateGrid failed!");

	//DebugGPU(grid, hip_stream);

	AdvectParticles<<<blocks2, threads2, 0, hip_stream>>> (grid, grid->B_term->device_alloc, grid->weighted_term->device_alloc);
	cuda_status = PostExecutionChecks(cuda_status, "AdvectParticles", false);
	CudaExceptionHandler(cuda_status, "AdvectParticles failed!");

	hipStreamSynchronize(hip_stream);

	//DebugGPU(grid, hip_stream);

	return cuda_status;
}