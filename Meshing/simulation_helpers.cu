#include "hip/hip_runtime.h"
#include "mpm.cuh"

__global__ void InitializeGrid(Grid* grid) {
	//Cell Boundaries
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z_bounds = blockIdx.z * blockDim.z + threadIdx.z;

	IndexPair incident(x_bounds, y_bounds, z_bounds);

	Cell* cell = new Cell();
	cell->velocity = Vector3D(0.0f, 0.0f, 0.0f);
	cell->mass = 0.0f;
	grid->AddCell(cell, incident.IX(grid->side_size_));

	for (size_t i = 0; i < grid->GetResolution(); i++) { //IDX, Particle
		Particle* particle = new Particle();
		particle->position = Vector3D(0.0f, 0.0f, 0.0f);
		particle->velocity = Vector3D(0.0f, 0.0f, 0.0f);
		particle->momentum = Matrix(3, 3, false);
		particle->mass = 0.0f;

		grid->AddParticle(particle, incident.IX(grid->side_size_) + i);
	}
}

__host__ hipError_t InitializeGridHost(Grid* grid) {
	for (size_t i = 0; i < grid->GetTotalSize(); i++) {
		Cell* cell = new Cell();
		cell->velocity = Vector3D(0.0f, 0.0f, 0.0f);
		cell->mass = 0.0f;
		grid->AddCell(cell, i);
	}


	for (size_t j = 0; j < grid->GetParticleCount(); j++) { //IDX, Particle
		Particle* particle = new Particle();
		particle->position = Vector3D(0.0f, 0.0f, 0.0f);
		particle->velocity = Vector3D(0.0f, 0.0f, 0.0f);
		particle->momentum = Matrix(3, 3, false);
		grid->AddParticle(particle, j);
	}
	return grid->DeviceTransfer(grid);
}

__global__ void UpdateGrid(Grid* grid) {
	//Cell Boundaries
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z_bounds = blockIdx.z * blockDim.z + threadIdx.z;

	IndexPair incident(x_bounds, y_bounds, z_bounds); //Current Index

	Cell* cell = grid->GetCell(incident); //Current Cell

	cell->velocity = cell->velocity / cell->mass; //Converting momentum to velocity

	//Applying gravity to velocity
	Vector3D gravity_vector(0.0f, 0.0f, grid->gravity);
	cell->velocity = cell->velocity + (gravity_vector * grid->dt);

	//Boundary Conditions
	if (x_bounds < 2 || x_bounds > grid->side_size_ - 3) {
		cell->velocity.dim[0] = 0;
	}
	if (y_bounds < 2 || y_bounds > grid->side_size_ - 3) {
		cell->velocity.dim[1] = 0;
	}
	if (z_bounds < 2 || z_bounds > grid->side_size_ - 3) {
		cell->velocity.dim[2] = 0;
	}
}

__device__ Vector3D* GetWeights(Vector3D cell_difference) { //Returns weights shared
	Vector3D weights[3]{}; //Array of weights

	weights[0] = (cell_difference.Negative() + 0.5f).Squared() * 0.5f;
	weights[1] = cell_difference.Squared().Negative() + 0.75f;
	weights[2] = (cell_difference + 0.5f).Squared() * 0.5f;

	return weights;
}

__device__ IndexPair* GetTraversals(IndexPair incident) {
	IndexPair incidents[27] = { incident, //D3Q27 tensor traversal
		incident.Left(), incident.Right(), incident.Front(), incident.Back(), incident.Up(), incident.Down(),
		incident.CornerLDownFront(), incident.CornerLDownBack(), incident.CornerRDownFront(), incident.CornerRDownBack(),
		incident.CornerLUpFront(), incident.CornerLUpBack(), incident.CornerRUpFront(), incident.CornerRUpBack(),
		incident.CornerLMidBack(), incident.CornerRMidBack(), incident.CornerLMidFront(), incident.CornerRMidFront(),
		incident.MidUpFront(), incident.MidUpBack(), incident.MidUpLeft(), incident.MidUpRight(),
		incident.MidDownFront() ,incident.MidDownBack(), incident.MidDownLeft(), incident.MidDownRight()
	};
	return incidents;
}

__global__ void ClearGrid(Grid* grid) {
	//Cell Boundaries
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z_bounds = blockIdx.z * blockDim.z + threadIdx.z;

	IndexPair incident(x_bounds, y_bounds, z_bounds); //Current position

	Cell* cell = grid->GetCell(incident);

	cell->mass = 0;
	cell->velocity.Reset();
}

__host__ hipError_t Grid::SimulateGPU(Grid* grid) {
	hipError_t cuda_status = hipSuccess;

	cuda_status = InitializeGridHost(grid);

	std::cout << "Allocated successfully " << grid->GetTotalSize() << " cells! (host)" << std::endl;
	std::cout << "Allocated successfully " << grid->GetParticleCount() << " particles! (host)" << std::endl;

	dim3 blocks, threads;
	unsigned int threads_per_dim = (unsigned int)cbrt (64);
	unsigned int block_count = ((grid->side_size_ + threads_per_dim) - 1) / (threads_per_dim);

	threads = dim3(threads_per_dim, threads_per_dim, threads_per_dim);
	blocks = dim3(block_count, block_count, block_count);

	dim3 blocks2, threads2;

	threads2 = dim3(threads_per_dim * grid->GetResolution(), threads_per_dim * grid->GetResolution(), threads_per_dim * grid->GetResolution());
	blocks2 = dim3(block_count * grid->GetResolution(), block_count * grid->GetResolution(), block_count * grid->GetResolution());

	InitializeGrid<<<blocks, threads>>> (grid->device_alloc_);
	cuda_status = PostExecutionChecks(cuda_status, "GridInitialization", true);

	std::cout << "Allocated successfully " << grid->GetTotalSize() << " cells! (device)" << std::endl;
	std::cout << "Allocated successfully " << grid->GetParticleCount() << " particles! (device)" << std::endl;

	UpdateCell<<<blocks, threads>>> (grid->device_alloc_);
	cuda_status = PostExecutionChecks(cuda_status, "CellMomentum", true);

	std::cout << "Ran through cell momentum!" << std::endl;

	SimulateGrid<<<blocks2, threads2>>> (grid->device_alloc_);
	cuda_status = PostExecutionChecks(cuda_status, "VelocityGradientSolve", true);

	std::cout << "Ran through the velocity gradient solve!" << std::endl;

	//UpdateGrid<<<blocks, threads>>> (grid->device_alloc_);
	cuda_status = PostExecutionChecks(cuda_status, "UpdateGrid", true);

	std::cout << "Updated the grid!" << std::endl;

	//AdvectParticles<<<blocks2, threads2>>> (grid->device_alloc_);
	cuda_status = PostExecutionChecks(cuda_status, "AdvectParticles", true);

	std::cout << "Advected the particles!" << std::endl;

	cuda_status = grid->HostTransfer();

	return cuda_status;
}