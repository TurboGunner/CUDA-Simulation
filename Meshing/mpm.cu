#include "mpm.cuh"

#include <iostream>

__host__ Grid::Grid(const Vector3D& sim_size_in, const float& resolution_in) { //Make sure these are integers!
	sim_size_ = sim_size_in;

	side_size_ = sim_size_.x();

	total_size_ = sim_size_.x() * sim_size_.y() * sim_size_.z();

	if (resolution_in <= 0.0f) {
		std::cout << "\n\nWarning! The resolution parameter should be a positive greater than 0.0. Set to default of 4 to prevent segfault." << std::endl;
		resolution_ = 4.0f;
	}
	else {
		resolution_ = resolution_in;
	}

	hipError_t cuda_status = hipSuccess;

	hipMalloc(&particles_device_, sizeof(Particle*) * total_size_ * resolution_);
	hipHostMalloc(&particles_, sizeof(Particle*) * total_size_ * resolution_);

	hipMalloc(&cells_device_, sizeof(Cell*) * total_size_);
	hipHostMalloc(&cells_, sizeof(Cell*) * total_size_);

	is_initialized_ = true;
}

__host__ void* Grid::operator new(size_t size) {
	void* ptr;
	hipHostMalloc(&ptr, sizeof(Grid));
	return ptr;
}

__host__ void Grid::operator delete(void* ptr) {
	free(ptr);
}

__host__ hipError_t Grid::DeviceTransfer(Grid*& src) {
	hipError_t cuda_status = hipSuccess;

	if (!device_allocated_status) {
		cuda_status = hipMalloc(&device_alloc_, sizeof(Grid));
		device_allocated_status = true;
		cuda_status = CopyFunction("DeviceTransferObject", device_alloc_, src, hipMemcpyHostToDevice, cuda_status, sizeof(Grid), 1);
	}
	return cuda_status;
}

__host__ hipError_t Grid::HostTransfer() {
	hipError_t cuda_status = hipSuccess;
	cuda_status = CopyFunction("HostTransferParticles", particles_, particles_device_, hipMemcpyDeviceToHost, cuda_status, sizeof(particles_), 1);
	cuda_status = CopyFunction("HostTransferCells", cells_, cells_device_, hipMemcpyDeviceToHost, cuda_status, sizeof(cells_), 1);
	cuda_status = hipDeviceSynchronize();

	return cuda_status;
}

__host__ __device__ size_t Grid::GetTotalSize() const {
#ifdef __CUDA_ARCH__
	return device_alloc_->total_size_;
#else
	return total_size_;
#endif
}

__host__ __device__ size_t Grid::GetParticleCount() const {
#ifdef __CUDA_ARCH__
	return device_alloc_->total_size_ * device_alloc_->resolution_;
#else
	return total_size_ * resolution_;
#endif
}

__host__ __device__ float Grid::GetResolution() const {
#ifdef __CUDA_ARCH__
	return device_alloc_->resolution_;
#else
	return resolution_;
#endif
}

__host__ __device__ void Grid::AddCell(Cell* cell, const size_t& index) {
	if (index >= GetTotalSize()) {
		printf("%s %d\n", "Warning! Out of bounds access (AddCell). Input Index: ", index);
	}
#ifdef __CUDA_ARCH__
	cells_device_[index] = cell;
#else
	cells_[index] = cell;
#endif
}

__host__ __device__ void Grid::AddParticle(Particle* particle, const size_t& index) {
	if (index >= GetParticleCount()) {
		printf("%s %d\n", "Warning! Out of bounds access (AddParticle). Input Index: ", index);
	}
#ifdef __CUDA_ARCH__
	particles_device_[index] = particle;
#else
	particles_[index] = particle;
#endif
}

__host__ __device__ Cell* Grid::GetCell(const size_t& index) {
	if (index >= total_size_) {
		printf("%s%zu\n", "Warning! Out of bounds access (size_t, Cell). Input Index: ", index);
	}
#ifdef __CUDA_ARCH__
	return cells_device_[index];
#else
	return cells_[index];
#endif
}

__host__ __device__ Cell* Grid::GetCell(IndexPair incident) {
	size_t index = incident.IX(side_size_);
	return GetCell(index);
}

__host__ __device__ Particle* Grid::GetParticle(const size_t& index) {
	if (index >= GetParticleCount()) {
		printf("%s %d\n", "Warning! Out of bounds access (size_t, Particle). Input Index: ", index);
	}
#ifdef __CUDA_ARCH__
	return particles_device_[index];
#else
	return particles_[index];
#endif
}

__host__ __device__ Particle* Grid::GetParticle(IndexPair& incident, const size_t& grid_offset) {
	size_t index = incident.IX(side_size_) * (grid_offset); //NOTE
	return GetParticle(index);
}