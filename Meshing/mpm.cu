#include "mpm.cuh"

#include <iostream>

__host__ Grid::Grid(const Vector3D& sim_size_in, const float resolution_in, const bool late_init_in) { //Make sure these are integers!
	sim_size_ = sim_size_in;

	side_size_ = sim_size_.x();

	total_size_ = sim_size_.x() * sim_size_.y() * sim_size_.z();

	if (resolution_in <= 0.0f) {
		std::cout << "\n\nWarning! The resolution parameter should be a positive greater than 0.0. Set to default of 4 to prevent segfault." << std::endl;
		resolution_ = 4.0f;
	}
	else {
		resolution_ = resolution_in;
	}

	late_init_ = late_init_in;

	hipError_t cuda_status = hipSuccess;
	
	if (!late_init_) {
		cuda_status = hipMalloc(&particle_position_device_, GetParticleCount() * sizeof(Vector3D));
		if (host_sync_) {
			cuda_status = hipHostMalloc(&particle_position_, GetParticleCount() * sizeof(Vector3D));
		}
	}

	cuda_status = hipMalloc(&particle_velocity_device_, GetParticleCount() * sizeof(Vector3D));
	cuda_status = hipMalloc(&particle_mass_device_, GetParticleCount() * sizeof(float));

	cuda_status = hipMalloc(&cell_velocity_device_, total_size_ * sizeof(Vector3D)); //NOTE
	//hipHostMalloc(&cells_, sizeof(Cell));
	cuda_status = hipMalloc(&cell_mass_device_, total_size_ * sizeof(float));

	momentum_matrices_ = Matrix::MatrixMassAllocation(GetParticleCount(), 3, 3);

	is_initialized_ = true;
}

__host__ Grid::~Grid() {
	hipError_t cuda_status = hipFree(particle_position_device_);
	cuda_status = hipFree(particle_velocity_device_);
	cuda_status = hipFree(particle_mass_device_);

	cuda_status = hipFree(cell_velocity_device_);
	cuda_status = hipFree(cell_mass_device_);

	if (host_sync_) {
		cuda_status = hipHostFree(particle_position_);
	}
	for (size_t i = 0; i < GetParticleCount(); i++) {
		momentum_matrices_[0].Destroy();
	}
}

__host__ void* Grid::operator new(size_t size) {
	void* ptr;
	hipHostMalloc(&ptr, sizeof(Grid));
	return ptr;
}

__host__ void Grid::operator delete(void* ptr) {
	free(ptr);
}

__host__ hipError_t Grid::DeviceTransfer(Grid*& src) {
	hipError_t cuda_status = hipSuccess;

	if (!device_allocated_status) {
		cuda_status = hipMalloc(&device_alloc_, sizeof(Grid));
		device_allocated_status = true;
		cuda_status = CopyFunction("DeviceTransferObject", device_alloc_, src, hipMemcpyHostToDevice, cuda_status, sizeof(Grid), 1);
	}
	return cuda_status;
}

__host__ hipError_t Grid::HostTransfer() { //NOTE
	hipError_t cuda_status = hipSuccess;
	cuda_status = CopyFunction("HostTransferParticlesPosition", particle_position_, particle_position_device_, hipMemcpyDeviceToHost, cuda_status, sizeof(Vector3D), GetParticleCount());
	cuda_status = hipDeviceSynchronize();

	return cuda_status;
}

__host__ __device__ size_t Grid::GetTotalSize() const {
#ifdef __CUDA_ARCH__
	return device_alloc_->total_size_;
#else
	return total_size_;
#endif
}

__host__ __device__ size_t Grid::GetParticleCount() const {
#ifdef __CUDA_ARCH__
	return device_alloc_->total_size_ * device_alloc_->resolution_;
#else
	return total_size_ * resolution_;
#endif
}

__host__ __device__ float Grid::GetResolution() const {
#ifdef __CUDA_ARCH__
	return device_alloc_->resolution_;
#else
	return resolution_;
#endif
}

__host__ __device__ Vector3D& Grid::GetVelocity(const size_t index) {
	assert(index <= GetParticleCount());
#ifdef __CUDA_ARCH__
	return particle_velocity_device_[index];
#else
	return particle_velocity_[index];
#endif
}

__host__ __device__ Vector3D& Grid::GetVelocity(IndexPair incident) {
	size_t index = incident.IX(side_size_);
	return GetVelocity(index);
}

__host__ __device__ Vector3D& Grid::GetPosition(const size_t index) {
	assert(index <= GetParticleCount());
#ifdef __CUDA_ARCH__
	return particle_position_device_[index];
#else
	return particle_position_[index];
#endif
}

__host__ __device__ Vector3D& Grid::GetPosition(IndexPair incident) {
	size_t index = incident.IX(side_size_);
	return GetPosition(index);
}

__host__ __device__ Matrix& Grid::GetMomentum(const size_t index) { //NOTE: DEVICE ONLY!
	assert(index <= GetParticleCount());
	return momentum_matrices_[index];
}

__host__ __device__ Matrix& Grid::GetMomentum(IndexPair incident) {
	size_t index = incident.IX(side_size_);
	return GetMomentum(index);
}

__host__ __device__ float& Grid::GetParticleMass(const size_t index) {
	assert(index <= GetParticleCount());
#ifdef __CUDA_ARCH__
	return particle_mass_device_[index];
#else
	return particle_mass_[index];
#endif
}

__host__ __device__ float& Grid::GetParticleMass(IndexPair incident) {
	size_t index = incident.IX(side_size_);
	return GetParticleMass(index);
}

__host__ __device__ float& Grid::GetCellMass(const size_t index) {
	assert(index <= total_size_);
#ifdef __CUDA_ARCH__
	return cell_mass_device_[index];
#else
	return cell_mass_[index];
#endif
}

__host__ __device__ float& Grid::GetCellMass(IndexPair incident) {
	size_t index = incident.IX(side_size_);
	return GetCellMass(index);
}

__host__ __device__ Vector3D& Grid::GetCellVelocity(const size_t index) {
	assert(index <= total_size_);
#ifdef __CUDA_ARCH__
	return cell_velocity_device_[index];
#else
	return cell_velocity_[index];
#endif
}

__host__ __device__ Vector3D& Grid::GetCellVelocity(IndexPair incident) {
	size_t index = incident.IX(side_size_);
	return GetCellVelocity(index);
}