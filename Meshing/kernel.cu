﻿
#include "hip/hip_runtime.h"
#include ""

#include "matrix.cuh"
#include "mpm.cuh"

#include "../CUDATest/handler_methods.hpp"
#include "../CUDATest/handler_classes.hpp"

#include <stdio.h>

#include <iostream>
#include <functional>
#include <vector>

using std::function;
using std::vector;

int main() {
    hipError_t cuda_status = hipSuccess;

    function<hipError_t()> set_device_func = []() { return hipSetDevice(0); };
    cuda_status = WrapperFunction(set_device_func, "hipSetDevice failed!", "main",
        cuda_status, "Do you have a CUDA-capable GPU installed?");

    //Matrix* matrix = Matrix::Create(4, 3, false);

    //Matrix::WeightedLeastSquares(matrix);

    size_t dim = 64;

    Grid* grid = new Grid(Vector3D(dim, dim, dim), 4, false);

    hipStream_t stream;

    cuda_status = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    size_t frames = 20;

    for (int i = 0; i < frames; i++) {
        cuda_status = Grid::SimulateGPU(grid, stream);
        CudaExceptionHandler(cuda_status, "GPU sim at " + std::to_string(frames) + " failed!");
    }

    cuda_status = hipDeviceReset();
    CudaExceptionHandler(cuda_status, "hipDeviceReset failed!");

    return 0;
}