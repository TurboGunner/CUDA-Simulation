﻿
#include "hip/hip_runtime.h"
#include ""

#include "matrix.cuh"
#include "mpm.cuh"

#include "../CUDATest/handler_methods.hpp"
#include "../CUDATest/handler_classes.hpp"

#include <stdio.h>

#include <iostream>
#include <functional>
#include <vector>

using std::function;
using std::vector;

int main() {
    hipError_t cuda_status = hipSuccess;

    function<hipError_t()> set_device_func = []() { return hipSetDevice(0); };
    cuda_status = WrapperFunction(set_device_func, "hipSetDevice failed!", "main",
        cuda_status, "Do you have a CUDA-capable GPU installed?");

    //Matrix* matrix = Matrix::Create(4, 3, false);

    //Matrix::WeightedLeastSquares(matrix);

    //size_t dim = 32;

    //Grid* grid = new Grid(Vector3D(dim, dim, dim), 4);

    //cuda_status = Grid::SimulateGPU(grid);

    //cuda_status = hipDeviceReset();
    CudaExceptionHandler(cuda_status, "hipDeviceReset failed!");

    return 0;
}