#include "hip/hip_runtime.h"
#include "fluid_sim_cuda.cuh"

__global__ void LinearSolverKernel(HashMap<IndexPair, float, HashDupe<IndexPair>>* data, HashMap<IndexPair, float, HashDupe<IndexPair>>* data_prev, float a_fac, float c_fac, unsigned int length, unsigned int iter, int bounds) {
	unsigned int y_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int x_bounds = blockIdx.y * blockDim.y + threadIdx.y;

	//printf("%u %u \n", y_bounds, x_bounds);
	if (threadIdx.x < length - 1 && threadIdx.y < length - 1) {
		auto* pairs = GetAdjacentCoordinates(IndexPair(y_bounds, x_bounds));
		printf("%d", pairs->size_);
		for (int i = 0; i < iter; i++) {
			printf("%f", pairs->Get(Direction::Origin).x);
			(*data)[pairs->Get(Direction::Origin)] = ((*data_prev)[pairs->Get(Direction::Origin)] +
				a_fac *
				(*data)[pairs->Get(Direction::Right)]
					+ (*data)[pairs->Get(Direction::Left)]
					+ (*data)[pairs->Get(Direction::Up)]
					+ (*data)[pairs->Get(Direction::Down)])
				* (1.0f / c_fac);
		}
		if (x_bounds * y_bounds >= (length * length)) {
			BoundaryConditions(bounds, data, length);
		}
	}
}

void LinearSolverCuda(int bounds, AxisData& current, AxisData& previous, const float& a_fac, const float& c_fac, const unsigned int& iter, const unsigned int& length) {
	unsigned int alloc_size = length * length;
	CudaMethodHandler handler(alloc_size, "LinearSolverKernel");

	hipError_t cuda_status = hipSuccess;

	dim3 blocks, threads;
	std::cout << "Length: " << length << std::endl;
	ThreadAllocator(blocks, threads, length);

	LinearSolverKernel<<<blocks, threads>>> (current.map_, previous.map_, a_fac, c_fac, length, iter, bounds);

	handler.PostExecutionChecks(cuda_status);
}