#include "hip/hip_runtime.h"
#include "fluid_sim_cuda.cuh"

__global__ void LinearSolverKernel(HashMap<float>* data, HashMap<float>* data_prev, float a_fac, float c_fac, uint3 length, unsigned int iter, int bounds) {
	unsigned int z_bounds = blockIdx.x * blockDim.x + threadIdx.x + 1;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y + 1;
	unsigned int x_bounds = blockIdx.z * blockDim.z + threadIdx.z + 1;

	if (threadIdx.x < length.x - 1 && threadIdx.y < length.y - 1 && threadIdx.z < length.z - 1) {
		IndexPair incident(z_bounds, y_bounds, x_bounds);
		for (int i = 0; i < iter; i++) {
			float compute = data_prev->Get(incident.IX(length.x)) +
				a_fac *
				(data->Get(incident.Right().IX(length.x))
					+ data->Get(incident.Left().IX(length.x))
					+ data->Get(incident.Up().IX(length.x))
					+ data->Get(incident.Down().IX(length.x)))
				* (1.0f / c_fac);
			data->Get(incident.IX(length.x)) = compute;
		}
	}
	if (x_bounds == length.x - 1 && y_bounds == length.y - 1 && z_bounds == length.z - 1) {
		BoundaryConditions(bounds, data, length);
	}
}

void LinearSolverCuda(int bounds, AxisData& current, AxisData& previous, const float& a_fac, const float& c_fac, const unsigned int& iter, const uint3& length) {
	hipError_t cuda_status = hipSuccess;

	dim3 blocks, threads;
	ThreadAllocator(blocks, threads, length.x);

	HashMap<float>* c_map = current.map_->device_alloc_,
		*p_map = previous.map_->device_alloc_;

	LinearSolverKernel<<<blocks, threads>>> (c_map, p_map, a_fac, c_fac, length, iter, bounds);

	cuda_status = PostExecutionChecks(cuda_status, "LinearSolverKernel");
}