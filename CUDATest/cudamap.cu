#include "hip/hip_runtime.h"
#include "cudamap.cuh"

#include <iostream>

HashMap::HashMap() {
    hash_table_size_ = DEFAULT_SIZE;
    Initialization();
}

__host__ __device__ HashMap::HashMap(const size_t& hash_table_size) {
    if (hash_table_size < 1) {
        printf("%s\n", "The input size for the hash table should be at least 1!");
    }
    hash_table_size_ = hash_table_size;
    Initialization();
}

__host__ __device__ void HashMap::Initialization() {
    hipMalloc(&table_, (size_t)sizeof(float) * hash_table_size_);
    hipMalloc(&hashes_, (size_t)sizeof(int) * hash_table_size_);

    hipHostMalloc(&table_host_, sizeof(float) * hash_table_size_);
    hipHostMalloc(&hashes_host_, (size_t)sizeof(int) * hash_table_size_);

    for (size_t i = 0; i < hash_table_size_; i++) {
        hashes_host_[i] = 0;
    }
}

__host__ __device__ size_t HashMap::hash_func_(IndexPair i1) {
    size_t hash = i1.x + (i1.y * cbrt(hash_table_size_)) + (i1.z * (cbrt(hash_table_size_) * cbrt(hash_table_size_)));
    return hash;
}

HashMap::~HashMap() {
    hipFree(table_);
    free((void*)table_host_);

    hipFree(hashes_);
    free((void*)hashes_host_);

    hipFree(device_alloc_);
}

__host__ __device__ void* HashMap::operator new(size_t size) {
    void* ptr;
    ptr = malloc(sizeof(HashMap));
    return ptr;
}

__host__ __device__ void HashMap::operator delete(void* ptr) {
    hipDeviceSynchronize();
    free(ptr);
}

__host__ __device__ long HashMap::FindHash(const int& hash) {
    if (hash > hash_table_size_) {
        return -1;
    }
#ifdef __CUDA_ARCH__
    return hashes_[hash] - 1;
#else
    return hashes_host_[hash] - 1;
#endif
}

__host__ __device__ float& HashMap::Get(const IndexPair& key) {
    size_t hash = hash_func_(key);
    long hash_pos = FindHash(hash);
#ifdef __CUDA_ARCH__
    return table_[hash_pos];
#else
    return table_host_[hash_pos];
#endif
}

__host__ __device__ float& HashMap::Get(const int& index) {
#ifdef __CUDA_ARCH__
    return table_[index];
#else
    if (index < 0) {
        throw std::invalid_argument("Invalid index!");
    }
    return table_host_[index];
#endif
}

__host__ __device__ void HashMap::Put(const IndexPair& key, const float& value) {
#ifdef __CUDA_ARCH__
#else
    size_t hash = hash_func_(key);
    long hash_pos = FindHash(hash);

    if (hash_pos == -1 && size_ <= hash_table_size_) {
        hashes_host_[hash] = size_ + 1;
        table_host_[size_] = value;
        size_++;
        return;
    }
#endif
#ifdef __CUDA_ARCH__
    printf("%f", key.IX(cbrt(hash_table_size_)));
    table_[key.IX(cbrt(hash_table_size_))] = value;
#else
    table_host_[hash_pos] = value;
#endif
}

__host__ __device__ void HashMap::Put(int key, float value) {
#ifdef __CUDA_ARCH__
    table_[key] = value;
#else
    table_host_[key] = value;
#endif
}

void HashMap::DeviceTransfer(hipError_t& cuda_status, HashMap*& src, HashMap*& ptr) {
    cuda_status = CopyFunction("DeviceTransferTable", table_, table_host_, hipMemcpyHostToDevice, cuda_status, sizeof(float), hash_table_size_);
    cuda_status = CopyFunction("DeviceTransferHash", hashes_, hashes_host_, hipMemcpyHostToDevice, cuda_status, sizeof(int), hash_table_size_);
    if (!device_allocated_status) {
        cuda_status = hipMalloc(&ptr, sizeof(HashMap));
        device_allocated_status = true;
        cuda_status = CopyFunction("DeviceTransferObject", ptr, src, hipMemcpyHostToDevice, cuda_status, sizeof(HashMap), 1);
        device_alloc_ = ptr;
    }
    else {
        ptr = device_alloc_;
    }
    std::cout << table_host_[0] << std::endl;
}

__host__ void HashMap::HostTransfer(hipError_t& cuda_status) {
    cuda_status = CopyFunction("HostTransferTable", table_host_, table_, hipMemcpyDeviceToHost, cuda_status, sizeof(float), hash_table_size_);
    hipDeviceSynchronize();
}

__host__ __device__ void HashMap::Remove(const IndexPair& key) {
    size_t hash = hash_func_(key);
    unsigned long hash_pos = FindHash(hash);
    if (hash_pos == -1) {
        return;
    }
    int hash_pos_empty = 0, hash_empty = -1;
#ifdef __CUDA_ARCH__
    table_[hash_pos] = hash_pos_empty;
    hashes_[hash] = hash_empty;
#else
    table_host_[hash_pos] = hash_pos_empty;
    hashes_host_[hash] = hash_empty;
#endif

    size_--;
}

__host__ __device__ float& HashMap::operator[](const IndexPair& key) {
    float output = Get(key);
    return output;
}

__host__ __device__ float& HashMap::operator[](const int& index) {
    float output = Get(index);
    return output;
}

HashMap& HashMap::operator=(const HashMap& src) {
    if (table_ == src.table_) {
        return *this;
    }
    table_ = src.table_;
    hashes_ = src.hashes_;
    return *this;
}

__host__ __device__ size_t HashMap::Size() const {
    return hash_table_size_;
}