#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include "fluid_sim_cuda.cuh"

#include "handler_methods.hpp"
#include "vector_field.hpp"
#include "fluid_sim.hpp"

#include <iostream>
#include <functional>

#include <stdio.h>

using std::vector;
using std::reference_wrapper;
using std::function;

int main()
{
    const float FPS_10 = .1f, FPS_24 = 0.04166f, FPS_60 = 0.0166f;

    unsigned int iter = 32, side_bound = 256;
    uint3 sim_dimensions;

    sim_dimensions.x = side_bound;
    sim_dimensions.y = side_bound;
    sim_dimensions.z = side_bound;

    FluidSim simulation(FPS_60, 1.0f, 1.0f, sim_dimensions, iter, 0.332f);

    hipError_t cuda_status = hipSuccess;

    function<hipError_t()> set_device_func = []() { return hipSetDevice(0); };
    cuda_status = WrapperFunction(set_device_func, "hipSetDevice failed!", "main",
        cuda_status, "Do you have a CUDA-capable GPU installed?");

    simulation.Simulate();

    cuda_status = hipDeviceReset();
    CudaExceptionHandler(cuda_status, "hipDeviceReset failed!");

    return 0;
}