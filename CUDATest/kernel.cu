#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "handler_methods.hpp"
#include "vector_field.hpp"

#include <iostream>

#include <stdio.h>

using std::vector;
using std::reference_wrapper;

hipError_t AddWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void AddKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] * b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 },
        b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel
    hipError_t cudaStatus = AddWithCuda(c, a, b, arraySize);
    CudaExceptionHandler(cudaStatus, "addWithCuda failed!");

    printf("{1,2,3,4,5} * {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    cudaStatus = hipDeviceReset();
    CudaExceptionHandler(cudaStatus, "hipDeviceReset failed!");

    VectorField field(3, 3);
    field.GetVectorMap()[IndexPair(2, 2)] = F_Vector(2, 2);
    std::cout << field.ToString() << std::endl;

    return 0;
}

hipError_t AddWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int* dev_a = nullptr, *dev_b = nullptr, *dev_c = nullptr;
    hipError_t cudaStatus;

    vector<reference_wrapper<int*>> bidoof;
    bidoof.insert(bidoof.end(), { dev_c, dev_a, dev_b });

    cudaStatus = hipSetDevice(0); //Assumes no multi-GPU
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    CudaMemoryAllocator(bidoof, (size_t) size, sizeof(int));
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = CopyFunction("hipMemcpy failed!", dev_a, a, hipMemcpyHostToDevice, cudaStatus, (size_t)size, sizeof(int));
    cudaStatus = CopyFunction("hipMemcpy failed!", dev_b, b, hipMemcpyHostToDevice, cudaStatus, (size_t)size, sizeof(int));

    // Launch a kernel on the GPU with one thread for each element.
    AddKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        CudaMemoryFreer(bidoof);
        return cudaStatus;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        CudaMemoryFreer(bidoof);
        return cudaStatus;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = CopyFunction("hipMemcpy failed!", c, dev_c, hipMemcpyDeviceToHost, cudaStatus, (size_t)size, sizeof(int));
    if (cudaStatus != hipSuccess) {
        CudaMemoryFreer(bidoof);
    }
    return cudaStatus;
}