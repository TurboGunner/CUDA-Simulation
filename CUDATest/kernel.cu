﻿#include "hip/hip_runtime.h"
#include ""

#include "fluid_sim_cuda.cuh"

#include "handler_methods.hpp"
#include "vector_field.hpp"
#include "fluid_sim.hpp"

#include <iostream>
#include <functional>

#include <stdio.h>

using std::vector;
using std::reference_wrapper;
using std::function;

int main()
{
    unsigned int iter = 32, side_bound = 128;
    FluidSim simulation(.1f, .1f, 1, side_bound, side_bound, iter);

    hipError_t cuda_status = hipSuccess;

    function<hipError_t()> set_device_func = []() { return hipSetDevice(0); };
    cuda_status = WrapperFunction(set_device_func, "hipSetDevice failed!", "main",
        cuda_status, "Do you have a CUDA-capable GPU installed?");

    simulation.Simulate();

    cuda_status = hipDeviceReset();
    CudaExceptionHandler(cuda_status, "hipDeviceReset failed!");

    return 0;
}