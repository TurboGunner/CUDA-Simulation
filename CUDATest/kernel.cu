#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "handler_methods.hpp"

#include <iostream>

#include <stdio.h>

using std::vector;
using std::reference_wrapper;

hipError_t AddWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void AddKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] * b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 },
        b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = AddWithCuda(c, a, b, arraySize);
    CudaExceptionHandler(cudaStatus, "addWithCuda failed!");

    printf("{1,2,3,4,5} * {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    cudaStatus = hipDeviceReset();

    CudaExceptionHandler(cudaStatus, "hipDeviceReset failed!");

    return 0;
}

hipError_t AddWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int* dev_a = nullptr, *dev_b = nullptr, *dev_c = nullptr;
    hipError_t cudaStatus;

    vector<reference_wrapper<int*>> bidoof;
    bidoof.insert(bidoof.end(), { dev_c, dev_a, dev_b });

    cudaStatus = hipSetDevice(0); //Assumes no multi-GPU
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)

    CudaMemoryAllocator(bidoof, (size_t) size);

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    AddKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    Error:
        CudaMemoryFreer(bidoof);
    return cudaStatus;
}