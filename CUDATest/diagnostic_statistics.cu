#include "hip/hip_runtime.h"
#include "diagnostic_statistics.cuh"

__global__ void Maximum(HashMap* data, uint3 length, float* max) {
	unsigned int z_bounds = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int x_bounds = blockIdx.z * blockDim.z + threadIdx.z;

	IndexPair incident(x_bounds, y_bounds, z_bounds);

	if (data->Get(incident.IX(length.x)) > *max) {
		*max = data->Get(incident.IX(length.x));
	}
}

float MaximumCuda(AxisData& map, const uint3& length) {
	hipError_t cuda_status = hipSuccess;

	float* max = nullptr,
		*max_copy = new float();

	dim3 blocks, threads;
	ThreadAllocator(blocks, threads, length.x);

	hipMalloc(&max, sizeof(float));
	cuda_status = CopyFunction("MaximumCudaDevice", max, max_copy, hipMemcpyHostToDevice, cuda_status, sizeof(float), 1);

	Maximum<<<blocks, threads>>> (map.map_->device_alloc_, length, max);

	cuda_status = CopyFunction("MaximumCudaDevice", max_copy, max, hipMemcpyDeviceToHost, cuda_status, sizeof(float), 1);

	cuda_status = PostExecutionChecks(cuda_status, "MaximumCudaKernel");

	float result = *max_copy;

	delete max_copy;
	hipFree(max);

	return result;
}