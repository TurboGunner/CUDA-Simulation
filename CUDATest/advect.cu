#include "hip/hip_runtime.h"
#include "fluid_sim_cuda.cuh"

__global__ void AdvectKernel(float* result_ptr, float* data, float* data_prev, float3* velocity, float dt, unsigned int length) {
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x + 1;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y + 1;

	float x_current, x_previous, y_current, y_previous;

	float x_dt = dt * (length - 2);
	float y_dt = dt * (length - 2);

	float velocity_x_curr, velocity_x_prev, velocity_y_curr, velocity_y_prev;

	float x_value, y_value;

	if (threadIdx.x < length - 1 && threadIdx.y < length - 1) {
		x_value = (float) x_bounds - (x_dt * velocity[IX(x_bounds, y_bounds + 1, length)].x);
		y_value = (float) y_bounds - (y_dt * velocity[IX(x_bounds, y_bounds + 1, length)].y);

		if (x_value < 0.5f) {
			x_value = 0.5f;
		}
		if (x_value > length + 0.5f) {
			x_value = length + 0.5f;
		}
		x_current = x_value;
		x_previous = x_current + 1.0f;
		if (y_value < 0.5f) {
			y_value = 0.5f;
		}
		if (y_value > length + 0.5f) {
			y_value = length + 0.5f;
		}
		y_current = y_value;
		y_previous = y_current + 1.0f;

		velocity_x_prev = x_value - x_current;
		velocity_x_curr = 1.0f - velocity_x_prev;
		velocity_y_prev = y_value - y_current;
		velocity_y_curr = 1.0f - velocity_y_prev;

		data[IX(x_bounds, y_bounds + 1, length)] =
			((data_prev[IX(unsigned int(x_current), unsigned int(y_current + 1), length)] * velocity_y_curr) +
				(data_prev[IX(unsigned int(x_current), int(y_previous + 1), length)] * velocity_y_prev) * velocity_x_curr) +
			((data_prev[IX(unsigned int(x_previous), unsigned int(y_current + 1), length)] * velocity_y_curr) +
				(data_prev[IX(unsigned int(x_previous), unsigned int(y_previous + 1), length)] * velocity_y_prev) * velocity_x_prev);
	}
	result_ptr[IX(x_bounds, y_bounds + 1, length)] = data[IX(x_bounds, y_bounds + 1, length)];
	if (x_bounds * y_bounds >= (length * length)) {
		PointerBoundaries(result_ptr, length);
	}
}

float* AdvectCuda(int bounds, VectorField& current, VectorField& previous, VectorField& velocity, const float& dt, const unsigned int& length) {
	float* curr_copy_ptr = nullptr, *prev_copy_ptr = nullptr;

	float* current_ptr = current.FlattenMapX(), //Maybe make current and previous part of the same vector to consolidate?
		*prev_ptr = previous.FlattenMapX();

	float3* v_ptr = velocity.FlattenMap(),
		*v_copy_ptr = nullptr;

	unsigned int alloc_size = length * length;

	float* result_ptr = new float[alloc_size],
		*result_copy_ptr = nullptr;

	vector<reference_wrapper<float*>> bidoof;
	bidoof.insert(bidoof.end(), { curr_copy_ptr, prev_copy_ptr, result_copy_ptr });

	CudaMemoryAllocator(bidoof, (size_t) alloc_size, sizeof(float));

	vector<reference_wrapper<float3*>> bidoof2;
	bidoof2.insert(bidoof2.end(), { v_copy_ptr } );

	//Maybe have more consolidation by auto-copying allocated pointers to the GPU
	CudaMemoryAllocator(bidoof2, (size_t) alloc_size, sizeof(float3));

	hipError_t cuda_status = hipSuccess;

	cuda_status = CopyFunction("hipMemcpy failed!", curr_copy_ptr, current_ptr,
		hipMemcpyHostToDevice, cuda_status, (size_t) alloc_size,
		sizeof(float));

	cuda_status = CopyFunction("hipMemcpy failed!", prev_copy_ptr, prev_ptr,
		hipMemcpyHostToDevice, cuda_status, (size_t) alloc_size,
		sizeof(float));

	cuda_status = CopyFunction("hipMemcpy failed!", v_copy_ptr, v_ptr,
		hipMemcpyHostToDevice, cuda_status, (size_t) alloc_size,
		sizeof(float3));

	dim3 blocks, threads;
	ThreadAllocator(blocks, threads, length);

	AdvectKernel<<<blocks, threads>>> (result_copy_ptr, curr_copy_ptr, prev_copy_ptr, v_copy_ptr, dt, length);

	function<hipError_t()> error_check_func = []() { return hipGetLastError(); };
	cuda_status = WrapperFunction(error_check_func, "hipGetLastError (kernel launch)", "LinearSolverKernel", cuda_status);

	function<hipError_t()> sync_func = []() { return hipDeviceSynchronize(); };
	cuda_status = WrapperFunction(sync_func, "hipDeviceSynchronize", "LinearSolverKernel", cuda_status);

	cuda_status = CopyFunction("hipMemcpy failed!", result_ptr, result_copy_ptr,
		hipMemcpyDeviceToHost, cuda_status, (size_t)alloc_size,
		sizeof(float));

	if (cuda_status != hipSuccess) {
		CudaMemoryFreer(bidoof);
	}
	return result_ptr;
}