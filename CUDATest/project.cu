#include "hip/hip_runtime.h"
#include "fluid_sim_cuda.cuh"

__global__ void ProjectKernel(float3* velocity, float* data, float* data_prev, unsigned int length, unsigned int iter, int bounds) {
	unsigned int y_bounds = blockIdx.x * blockDim.x + threadIdx.x + 1;
	unsigned int x_bounds = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if (threadIdx.x < length - 1 && threadIdx.y < length - 1) {
		data[IX(x_bounds, y_bounds + 1, length)] =
			((velocity[IX(x_bounds + 1, y_bounds + 1, length)].x
				- velocity[IX(x_bounds - 1, y_bounds + 1, length)].x
				+ velocity[IX(x_bounds, y_bounds + 2, length)].y
				- velocity[IX(x_bounds, y_bounds, length)].y)
				* -0.5f) * (1.0f / length);
		//data_prev[IX(x_bounds, y_bounds + 1, length)] = 0;
	}
	if (x_bounds * y_bounds >= (length * length)) {
		PointerBoundaries(data, length);
		PointerBoundaries(data_prev, length);
		LinearSolverGPU(data, data_prev, 1, 4, length, iter, bounds);
	}

	if (threadIdx.x < length - 1 && threadIdx.y < length - 1) {
		velocity[IX(x_bounds, y_bounds + 1, length)].x -= 0.5f
			* (data_prev[IX(x_bounds + 1, y_bounds + 1, length)]
			- data_prev[IX(x_bounds - 1, y_bounds + 1, length)])
			* length;
		velocity[IX(x_bounds, y_bounds + 1, length)].y -= 0.5f
			* (data_prev[IX(x_bounds, y_bounds + 2, length)]
			- data_prev[IX(x_bounds, y_bounds, length)])
			* length;
	}
	if (x_bounds * y_bounds >= (length * length)) {
		if (bounds == 0) {
			PointerBoundariesVector(velocity, length);
		}
		else {
			PointerBoundariesSpecial(velocity, length);
		}
	}
}

void ProjectCuda(int bounds, VectorField& velocity, VectorField& velocity_prev, const unsigned int& length, const unsigned int& iter) {
	unsigned int alloc_size = length * length;
	CudaMethodHandler handler(alloc_size, "ProjectCudaKernel");

	float3* v_ptr = velocity.FlattenMap(), * v_copy_ptr = nullptr;
	float* v_x_prev_ptr = velocity_prev.FlattenMapX(), * v_x_prev_copy_ptr = nullptr;
	float* v_y_prev_ptr = velocity_prev.FlattenMapY(), * v_y_prev_copy_ptr = nullptr;

	handler.float_copy_ptrs_.insert(handler.float_copy_ptrs_.end(), { v_x_prev_copy_ptr, v_y_prev_copy_ptr });
	handler.float_ptrs_.insert(handler.float_ptrs_.end(), { v_x_prev_ptr, v_y_prev_ptr });

	handler.float3_copy_ptrs_.insert(handler.float3_copy_ptrs_.end(), { v_copy_ptr });
	handler.float3_ptrs_.insert(handler.float3_ptrs_.end(), { v_ptr });

	handler.AllocateCopyPointers();

	hipError_t cuda_status = hipSuccess;

	cuda_status = handler.CopyToMemory(hipMemcpyHostToDevice, cuda_status);

	dim3 blocks, threads;
	ThreadAllocator(blocks, threads, length);

	ProjectKernel<<<blocks, threads>>> (v_copy_ptr, v_x_prev_copy_ptr, v_y_prev_copy_ptr, length, iter, bounds);

	handler.PostExecutionChecks(cuda_status);

	cuda_status = CopyFunction("hipMemcpy failed at v_x_ptr!", v_x_prev_ptr, v_x_prev_copy_ptr,
		hipMemcpyDeviceToHost, cuda_status, (size_t)alloc_size,
		sizeof(float));

	cuda_status = CopyFunction("hipMemcpy failed at v_y_ptr!", v_y_prev_ptr, v_y_prev_copy_ptr,
		hipMemcpyDeviceToHost, cuda_status, (size_t)alloc_size,
		sizeof(float));

	float3* ptr = new float3[alloc_size];

	cuda_status = CopyFunction("hipMemcpy failed at v_ptr!", ptr, v_copy_ptr,
		hipMemcpyDeviceToHost, cuda_status, (size_t)alloc_size,
		sizeof(float3));

	velocity.RepackMapVector(ptr);
	velocity_prev.RepackMap(v_x_prev_ptr, v_y_prev_ptr);

	handler.float3_ptrs_.insert(handler.float3_ptrs_.end(), { ptr });
	std::cout << v_y_prev_ptr[1] << std::endl;
	handler.~CudaMethodHandler();
	std::cout << velocity_prev.GetVectorMap()[IndexPair(1, 1)].ToString() << std::endl;
}