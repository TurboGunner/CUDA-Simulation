#include "hip/hip_runtime.h"
#include "fluid_sim_cuda.cuh"

__global__ void ProjectKernel(float3* result_ptr, float* data, float* data_prev, float3* velocity, unsigned int length, unsigned int iter, int bounds) {
	unsigned int x_bounds = blockIdx.x * blockDim.x + threadIdx.x + 1;
	unsigned int y_bounds = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if (threadIdx.x < length - 1 && threadIdx.y < length - 1) {
		data[IX(x_bounds, y_bounds + 1, length)] =
			(velocity[IX(x_bounds + 1, y_bounds + 1, length)].x
				- velocity[IX(x_bounds - 1, y_bounds + 1, length)].x
				+ velocity[IX(x_bounds, y_bounds + 2, length)].y
				- velocity[IX(x_bounds, y_bounds, length)].y)
			* -0.5f * (1.0f / length);

		data_prev[IX(x_bounds, y_bounds + 1, length)] = 0;
	}
	printf("%.5f\n", data[IX(x_bounds, y_bounds + 1, length)]);
	if (x_bounds * y_bounds >= (length * length)) {
		PointerBoundaries(data, length);
		PointerBoundaries(data_prev, length);
		LinearSolverGPU(data, data_prev, 1, 4, length, iter, bounds);
	}

	if (threadIdx.x < length - 1 && threadIdx.y < length - 1) {
		result_ptr[IX(x_bounds, y_bounds + 1, length)].x -= 0.5f
			* (data_prev[IX(x_bounds + 1, y_bounds + 1, length)]
			- data_prev[IX(x_bounds - 1, y_bounds + 1, length)]) 
			* length;
		result_ptr[IX(x_bounds, y_bounds + 1, length)].y -= 0.5f
			* (data_prev[IX(x_bounds, y_bounds + 2, length)]
			- data_prev[IX(x_bounds, y_bounds, length)])
			* length;
	}
	if (x_bounds * y_bounds >= (length * length)) {
		PointerBoundariesSpecial(velocity, length);
	}
}

tuple<float3*, float*, float*> ProjectCuda(int bounds, VectorField& current, VectorField& previous, VectorField& velocity, const unsigned int& length, const unsigned int& iter) {
	float* curr_copy_ptr = nullptr, * prev_copy_ptr = nullptr;

	float* current_ptr = current.FlattenMapX(), //Maybe make current and previous part of the same vector to consolidate?
		* prev_ptr = previous.FlattenMapX();

	float3* v_ptr = velocity.FlattenMap(),
		* v_copy_ptr = nullptr;

	unsigned int alloc_size = length * length;

	float3* result_ptr = new float3[alloc_size],
		*result_copy_ptr = nullptr;

	vector<reference_wrapper<float*>> bidoof;
	bidoof.insert(bidoof.end(), { curr_copy_ptr, prev_copy_ptr });

	CudaMemoryAllocator(bidoof, (size_t)alloc_size, sizeof(float));

	vector<reference_wrapper<float3*>> bidoof2;
	bidoof2.insert(bidoof2.end(), { v_copy_ptr, result_copy_ptr });

	//Maybe have more consolidation by auto-copying allocated pointers to the GPU
	CudaMemoryAllocator(bidoof2, (size_t)alloc_size, sizeof(float3));

	hipError_t cuda_status = hipSuccess;

	cuda_status = CopyFunction("hipMemcpy failed!", curr_copy_ptr, current_ptr,
		hipMemcpyHostToDevice, cuda_status, (size_t)alloc_size,
		sizeof(float));

	cuda_status = CopyFunction("hipMemcpy failed!", prev_copy_ptr, prev_ptr,
		hipMemcpyHostToDevice, cuda_status, (size_t)alloc_size,
		sizeof(float));

	cuda_status = CopyFunction("hipMemcpy failed!", v_copy_ptr, v_ptr,
		hipMemcpyHostToDevice, cuda_status, (size_t)alloc_size,
		sizeof(float3));

	dim3 blocks, threads;
	ThreadAllocator(blocks, threads, length);

	ProjectKernel<<<blocks, threads>>> (result_copy_ptr, curr_copy_ptr, prev_copy_ptr, v_copy_ptr, length, iter, bounds);

	function<hipError_t()> error_check_func = []() { return hipGetLastError(); };
	cuda_status = WrapperFunction(error_check_func, "hipGetLastError (kernel launch)", "LinearSolverKernel", cuda_status);

	function<hipError_t()> sync_func = []() { return hipDeviceSynchronize(); };
	cuda_status = WrapperFunction(sync_func, "hipDeviceSynchronize", "LinearSolverKernel", cuda_status);

	cuda_status = CopyFunction("hipMemcpy failed!", result_ptr, result_copy_ptr,
		hipMemcpyDeviceToHost, cuda_status, (size_t)alloc_size,
		sizeof(float3));

	cuda_status = CopyFunction("hipMemcpy failed!", prev_ptr, prev_copy_ptr,
		hipMemcpyDeviceToHost, cuda_status, (size_t)alloc_size,
		sizeof(float));

	cuda_status = CopyFunction("hipMemcpy failed!", current_ptr, curr_copy_ptr,
		hipMemcpyDeviceToHost, cuda_status, (size_t)alloc_size,
		sizeof(float));

	if (cuda_status != hipSuccess) {
		CudaMemoryFreer(bidoof);
		CudaMemoryFreer(bidoof2);
	}

	return std::make_tuple(result_ptr, current_ptr, prev_ptr);
}