#include "hip/hip_runtime.h"
#include "fluid_sim_cuda.cuh"

__global__ void ProjectKernel(HashMap<float>* velocity_x, HashMap<float>* velocity_y, HashMap<float>* data, HashMap<float>* data_prev, unsigned int length, int bounds) {
	unsigned int y_bounds = blockIdx.x * blockDim.x + threadIdx.x + 1;
	unsigned int x_bounds = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if (threadIdx.x < length - 1 && threadIdx.y < length - 1) { 
		IndexPair incident(y_bounds, x_bounds);
		data->Get(incident.IX(length)) =
			((velocity_x->Get(incident.Right().IX(length))
				- velocity_x->Get(incident.Left().IX(length))
				+ velocity_y->Get(incident.Up().IX(length))
				- velocity_y->Get(incident.Down().IX(length)))
				* -0.5f) / length;
		data_prev->Get(incident.IX(length)) = 0;
	}
	if (x_bounds * y_bounds >= (length * length)) {
		BoundaryConditions(0, data, length);
		BoundaryConditions(0, data_prev, length);
	}
}

__global__ void ProjectKernel2(HashMap<float>* velocity_x, HashMap<float>* velocity_y, HashMap<float>* data, HashMap<float>* data_prev, unsigned int length, int bounds) {
	unsigned int y_bounds = blockIdx.x * blockDim.x + threadIdx.x + 1;
	unsigned int x_bounds = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if (threadIdx.x < length - 1 && threadIdx.y < length - 1) {
		IndexPair incident(y_bounds, x_bounds);
		float compute_x = velocity_x->Get(incident.IX(length)) - (-0.5f *
			(data_prev->Get(incident.Right().IX(length))
				- data_prev->Get(incident.Left().IX(length)))
			* length);

		float compute_y = velocity_y->Get(incident.IX(length)) - (-0.5f *
			(data_prev->Get(incident.Up().IX(length))
				- data_prev->Get(incident.Down().IX(length)))
			* length);
		velocity_x->Put(incident.IX(length), compute_x);
		velocity_y->Put(incident.IX(length), compute_y);
	}
	if (x_bounds * y_bounds >= (length * length)) {
		BoundaryConditions(bounds, velocity_x, length);
		BoundaryConditions(bounds, velocity_y, length);
	}
}

void ProjectCuda(int bounds, VectorField& velocity, VectorField& velocity_prev, const unsigned int& length, const unsigned int& iter) {
	unsigned int alloc_size = length * length;

	hipError_t cuda_status = hipSuccess;

	dim3 blocks, threads;
	ThreadAllocator(blocks, threads, length);

	HashMap<float>* v_map_x = nullptr, *v_map_y = nullptr,
		* x_map = nullptr, *y_map = nullptr;

	velocity.GetVectorMap()[0].map_->DeviceTransfer(cuda_status, velocity.GetVectorMap()[0].map_, v_map_x);
	velocity.GetVectorMap()[1].map_->DeviceTransfer(cuda_status, velocity.GetVectorMap()[1].map_, v_map_y);
	velocity_prev.GetVectorMap()[0].map_->DeviceTransfer(cuda_status, velocity_prev.GetVectorMap()[0].map_, x_map);
	velocity_prev.GetVectorMap()[1].map_->DeviceTransfer(cuda_status, velocity_prev.GetVectorMap()[1].map_, y_map);

	ProjectKernel<<<blocks, threads>>> (v_map_x, v_map_y, y_map, x_map, length, bounds);
	LinearSolverKernel<<<blocks, threads>>> (y_map, x_map, 1, 4, length, iter, bounds);
	ProjectKernel2<<<blocks, threads>>> (v_map_x, v_map_y, y_map, x_map, length, bounds);

	std::cout << "Yo Pierre, you wanna come out here? *door squeaking noise*" << std::endl;

	PostExecutionChecks(cuda_status, "ProjectCudaKernel");

	velocity.GetVectorMap()[0].map_->HostTransfer(cuda_status);
	velocity.GetVectorMap()[1].map_->HostTransfer(cuda_status);

	velocity.GetVectorMap()[0].map_->HostTransfer(cuda_status);
	velocity_prev.GetVectorMap()[1].map_->HostTransfer(cuda_status);
}