#include "hip/hip_runtime.h"
#include "fluid_sim_cuda.cuh"

__global__ void ProjectKernel(HashMap<IndexPair, F_Vector, Hash<IndexPair>>* velocity, HashMap<IndexPair, float, HashDupe<IndexPair>>* data, HashMap<IndexPair, float, HashDupe<IndexPair>>* data_prev, unsigned int length, unsigned int iter, int bounds) {
	unsigned int y_bounds = blockIdx.x * blockDim.x + threadIdx.x + 1;
	unsigned int x_bounds = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if (threadIdx.x < length - 1 && threadIdx.y < length - 1) {
		IndexPair incident(y_bounds, x_bounds);
		(*data)[incident] =
			((*velocity)[incident.Right()].vx_
				- (*velocity)[incident.Left()].vx_
				+ (*velocity)[incident.Up()].vy_
				- (*velocity)[incident.Down()].vy_
				* -0.5f) * (1.0f / length);
		(*data_prev)[IndexPair(y_bounds, x_bounds)] = 0;
	}
	if (x_bounds * y_bounds >= (length * length)) {
		BoundaryConditions(0, data, length);
		BoundaryConditions(0, data_prev, length);
		LinearSolverGPU(data, data_prev, 1, 4, length, iter, bounds);
	}

	if (threadIdx.x < length - 1 && threadIdx.y < length - 1) {
		IndexPair incident(y_bounds, x_bounds);
		(*velocity)[incident].vx_ -= 0.5f
			* ((*data_prev)[incident.Right()]
			- (*data_prev)[incident.Left()])
			* length;
		(*velocity)[incident].vy_ -= 0.5f
			* ((*data_prev)[incident.Up()]
			- (*data_prev)[incident.Down()])
			* length;
	}
	if (x_bounds * y_bounds >= (length * length)) {
		BoundaryConditions(bounds, velocity, length);
	}
}

void ProjectCuda(int bounds, VectorField& velocity, VectorField& velocity_prev, const unsigned int& length, const unsigned int& iter) {
	unsigned int alloc_size = length * length;
	CudaMethodHandler handler(alloc_size, "ProjectCudaKernel");

	hipError_t cuda_status = hipSuccess;

	dim3 blocks, threads;
	ThreadAllocator(blocks, threads, length);

	AxisData v_prev_x(length, Axis::X), v_prev_y(length, Axis::Y);

	velocity_prev.DataConstrained(Axis::X, v_prev_x);
	velocity_prev.DataConstrained(Axis::Y, v_prev_y);

	ProjectKernel<<<blocks, threads>>> (velocity.GetVectorMap(), v_prev_x.map_, v_prev_y.map_, length, iter, bounds);

	handler.PostExecutionChecks(cuda_status);
}